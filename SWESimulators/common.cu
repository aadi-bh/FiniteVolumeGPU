#include "hip/hip_runtime.h"
/*
This OpenCL kernel implements the Kurganov-Petrova numerical scheme 
for the shallow water equations, described in 
A. Kurganov & Guergana Petrova
A Second-Order Well-Balanced Positivity Preserving Central-Upwind
Scheme for the Saint-Venant System Communications in Mathematical
Sciences, 5 (2007), 133-160. 

Copyright (C) 2016  SINTEF ICT

This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

/**
  * Location of thread in block
  */
inline __device__ int get_local_id(int dim) {
    switch(dim) {
        case 0: return threadIdx.x; 
        case 1: return threadIdx.y;
        case 2: return threadIdx.z;
        default: return -1;
    }
}


/**
  * Get block index
  */
__device__ int get_group_id(int dim) {
    switch(dim) {
        case 0: return blockIdx.x;
        case 1: return blockIdx.y;
        case 2: return blockIdx.z;
        default: return -1;
    }
}

/**
  * Location of thread in global domain
  */
__device__ int get_global_id(int dim) {
    switch(dim) {
        case 0: return blockDim.x*blockIdx.x + threadIdx.x;
        case 1: return blockDim.y*blockIdx.y + threadIdx.y;
        case 2: return blockDim.z*blockIdx.z + threadIdx.z;
        default: return -1;
    }
}


__device__ int get_local_size(int dim) {
    switch(dim) {
        case 0: return blockDim.x;
        case 1: return blockDim.y;
        case 2: return blockDim.z;
        default: return -1;
    }
}



/**
  * Float3 operators 
  */
inline __device__ float3 operator*(const float a, const float3 b) {
    return make_float3(a*b.x, a*b.y, a*b.z);
}

inline __device__ float3 operator/(const float3 a, const float b) {
    return make_float3(a.x/b, a.y/b, a.z/b);
}

inline __device__ float3 operator-(const float3 a, const float3 b) {
    return make_float3(a.x-b.x, a.y-b.y, a.z-b.z);
}

inline __device__ float3 operator+(const float3 a, const float3 b) {
    return make_float3(a.x+b.x, a.y+b.y, a.z+b.z);
}


inline __device__ __host__ float clamp(const float f, const float a, const float b) {
    return fmaxf(a, fminf(f, b));
}





/**
  * Reads a block of data  with one ghost cell for the shallow water equations
  */
__device__ void readBlock1(float* h_ptr_, int h_pitch_,
                float* hu_ptr_, int hu_pitch_,
                float* hv_ptr_, int hv_pitch_,
                float Q[3][block_height+2][block_width+2], 
                const int nx_, const int ny_) {
    //Index of thread within block
    const int tx = get_local_id(0);
    const int ty = get_local_id(1);
    
    //Index of block within domain
    const int bx = get_local_size(0) * get_group_id(0);
    const int by = get_local_size(1) * get_group_id(1);
    
    //Read into shared memory
    for (int j=ty; j<block_height+2; j+=get_local_size(1)) {
        const int l = clamp(by + j, 0, ny_+1); // Out of bounds
        
        //Compute the pointer to current row in the arrays
        float* const h_row  = (float*) ((char*) h_ptr_  + h_pitch_*l);
        float* const hu_row = (float*) ((char*) hu_ptr_ + hu_pitch_*l);
        float* const hv_row = (float*) ((char*) hv_ptr_ + hv_pitch_*l);
        
        for (int i=tx; i<block_width+2; i+=get_local_size(0)) {
            const int k = clamp(bx + i, 0, nx_+1); // Out of bounds
            
            Q[0][j][i] = h_row[k];
            Q[1][j][i] = hu_row[k];
            Q[2][j][i] = hv_row[k];
        }
    }
}





/**
  * Reads a block of data  with two ghost cells for the shallow water equations
  */
__device__ void readBlock2(float* h_ptr_, int h_pitch_,
                float* hu_ptr_, int hu_pitch_,
                float* hv_ptr_, int hv_pitch_,
                float Q[3][block_height+4][block_width+4], 
                const int nx_, const int ny_) {
    //Index of thread within block
    const int tx = get_local_id(0);
    const int ty = get_local_id(1);
    
    //Index of block within domain
    const int bx = get_local_size(0) * get_group_id(0);
    const int by = get_local_size(1) * get_group_id(1);
    
    //Read into shared memory
    for (int j=ty; j<block_height+4; j+=get_local_size(1)) {
        const int l = clamp(by + j, 0, ny_+3); // Out of bounds
        
        //Compute the pointer to current row in the arrays
        float* const h_row  = (float*) ((char*) h_ptr_  + h_pitch_*l);
        float* const hu_row = (float*) ((char*) hu_ptr_ + hu_pitch_*l);
        float* const hv_row = (float*) ((char*) hv_ptr_ + hv_pitch_*l);
        
        for (int i=tx; i<block_width+4; i+=get_local_size(0)) {
            const int k = clamp(bx + i, 0, nx_+3); // Out of bounds
            
            Q[0][j][i] = h_row[k];
            Q[1][j][i] = hu_row[k];
            Q[2][j][i] = hv_row[k];
        }
    }
}




/**
  * Writes a block of data to global memory for the shallow water equations.
  */
__device__ void writeBlock1(float* h_ptr_, int h_pitch_,
                 float* hu_ptr_, int hu_pitch_,
                 float* hv_ptr_, int hv_pitch_,
                 float Q[3][block_height+2][block_width+2],
                 const int nx_, const int ny_) {
    //Index of thread within block
    const int tx = get_local_id(0);
    const int ty = get_local_id(1);
    
    //Index of cell within domain
    const int ti = get_global_id(0) + 1; //Skip global ghost cells, i.e., +1
    const int tj = get_global_id(1) + 1;
    
    //Only write internal cells
    if (ti > 0 && ti < nx_+1 && tj > 0 && tj < ny_+1) {
        const int i = tx + 1; //Skip local ghost cells, i.e., +1
        const int j = ty + 1;

        float* const h_row  = (float*) ((char*) h_ptr_  + h_pitch_*tj);
        float* const hu_row = (float*) ((char*) hu_ptr_ + hu_pitch_*tj);
        float* const hv_row = (float*) ((char*) hv_ptr_ + hv_pitch_*tj);
        
        h_row[ti]  = Q[0][j][i];
        hu_row[ti] = Q[1][j][i];
        hv_row[ti] = Q[2][j][i];
    }
}





/**
  * Writes a block of data to global memory for the shallow water equations.
  */
__device__ void writeBlock2(float* h_ptr_, int h_pitch_,
                 float* hu_ptr_, int hu_pitch_,
                 float* hv_ptr_, int hv_pitch_,
                 float Q[3][block_height+4][block_width+4], 
                 const int nx_, const int ny_) {
    //Index of thread within block
    const int tx = get_local_id(0);
    const int ty = get_local_id(1);
    
    //Index of cell within domain
    const int ti = get_global_id(0) + 2; //Skip global ghost cells, i.e., +2
    const int tj = get_global_id(1) + 2;
    
    //Only write internal cells
    if (ti > 1 && ti < nx_+2 && tj > 1 && tj < ny_+2) {
        const int i = tx + 2; //Skip local ghost cells, i.e., +2
        const int j = ty + 2;

        float* const h_row  = (float*) ((char*) h_ptr_ + h_pitch_*tj);
        float* const hu_row = (float*) ((char*) hu_ptr_ + hu_pitch_*tj);
        float* const hv_row = (float*) ((char*) hv_ptr_ + hv_pitch_*tj);
        
        h_row[ti]  = Q[0][j][i];
        hu_row[ti] = Q[1][j][i];
        hv_row[ti] = Q[2][j][i];
    }
}






/**
  * No flow boundary conditions for the shallow water equations
  * with one ghost cell in each direction
  */
__device__ void noFlowBoundary1(float Q[3][block_height+2][block_width+2], const int nx_, const int ny_) {
    //Global index
    const int ti = get_global_id(0) + 1; //Skip global ghost cells, i.e., +1
    const int tj = get_global_id(1) + 1;
    
    //Block-local indices
    const int tx = get_local_id(0);
    const int ty = get_local_id(1);
    
    const int i = tx + 1; //Skip local ghost cells, i.e., +1
    const int j = ty + 1;
    
    //Fix boundary conditions
    if (ti == 1) {
        Q[0][j][i-1] =  Q[0][j][i];
        Q[1][j][i-1] = -Q[1][j][i];
        Q[2][j][i-1] =  Q[2][j][i];
    }
    if (ti == nx_) {
        Q[0][j][i+1] =  Q[0][j][i];
        Q[1][j][i+1] = -Q[1][j][i];
        Q[2][j][i+1] =  Q[2][j][i];
    }
    if (tj == 1) {
        Q[0][j-1][i] =  Q[0][j][i];
        Q[1][j-1][i] =  Q[1][j][i];
        Q[2][j-1][i] = -Q[2][j][i];
    }
    if (tj == ny_) {
        Q[0][j+1][i] =  Q[0][j][i];
        Q[1][j+1][i] =  Q[1][j][i];
        Q[2][j+1][i] = -Q[2][j][i];
    }
}






/**
  * No flow boundary conditions for the shallow water equations
  * with two ghost cells in each direction
  */
__device__ void noFlowBoundary2(float Q[3][block_height+4][block_width+4], const int nx_, const int ny_) {
    //Global index
    const int ti = get_global_id(0) + 2; //Skip global ghost cells, i.e., +2
    const int tj = get_global_id(1) + 2;
    
    //Block-local indices
    const int tx = get_local_id(0);
    const int ty = get_local_id(1);
    
    const int i = tx + 2; //Skip local ghost cells, i.e., +2
    const int j = ty + 2;
    
    if (ti == 2) {
        Q[0][j][i-1] =  Q[0][j][i];
        Q[1][j][i-1] = -Q[1][j][i];
        Q[2][j][i-1] =  Q[2][j][i];
        
        Q[0][j][i-2] =  Q[0][j][i+1];
        Q[1][j][i-2] = -Q[1][j][i+1];
        Q[2][j][i-2] =  Q[2][j][i+1];
    }
    if (ti == nx_+1) {
        Q[0][j][i+1] =  Q[0][j][i];
        Q[1][j][i+1] = -Q[1][j][i];
        Q[2][j][i+1] =  Q[2][j][i];
        
        Q[0][j][i+2] =  Q[0][j][i-1];
        Q[1][j][i+2] = -Q[1][j][i-1];
        Q[2][j][i+2] =  Q[2][j][i-1];
    }
    if (tj == 2) {
        Q[0][j-1][i] =  Q[0][j][i];
        Q[1][j-1][i] =  Q[1][j][i];
        Q[2][j-1][i] = -Q[2][j][i];
        
        Q[0][j-2][i] =  Q[0][j+1][i];
        Q[1][j-2][i] =  Q[1][j+1][i];
        Q[2][j-2][i] = -Q[2][j+1][i];
    }
    if (tj == ny_+1) {
        Q[0][j+1][i] =  Q[0][j][i];
        Q[1][j+1][i] =  Q[1][j][i];
        Q[2][j+1][i] = -Q[2][j][i];
        
        Q[0][j+2][i] =  Q[0][j-1][i];
        Q[1][j+2][i] =  Q[1][j-1][i];
        Q[2][j+2][i] = -Q[2][j-1][i];
    }
}






/**
  * Evolves the solution in time along the x axis (dimensional splitting)
  */
__device__ void evolveF1(float Q[3][block_height+2][block_width+2],
              float F[3][block_height+1][block_width+1],
              const int nx_, const int ny_,
              const float dx_, const float dt_) {
    //Index of thread within block
    const int tx = get_local_id(0);
    const int ty = get_local_id(1);
    
    //Index of cell within domain
    const int ti = get_global_id(0) + 1; //Skip global ghost cells, i.e., +1
    const int tj = get_global_id(1) + 1;
    
    if (ti > 0 && ti < nx_+1 && tj > 0 && tj < ny_+1) {
        const int i = tx + 1; //Skip local ghost cells, i.e., +1
        const int j = ty + 1;
        
        Q[0][j][i] = Q[0][j][i] + (F[0][ty][tx] - F[0][ty][tx+1]) * dt_ / dx_;
        Q[1][j][i] = Q[1][j][i] + (F[1][ty][tx] - F[1][ty][tx+1]) * dt_ / dx_;
        Q[2][j][i] = Q[2][j][i] + (F[2][ty][tx] - F[2][ty][tx+1]) * dt_ / dx_;
    }
}






/**
  * Evolves the solution in time along the x axis (dimensional splitting)
  */
__device__ void evolveF2(float Q[3][block_height+4][block_width+4],
              float F[3][block_height+1][block_width+1],
              const int nx_, const int ny_,
              const float dx_, const float dt_) {
    //Index of thread within block
    const int tx = get_local_id(0);
    const int ty = get_local_id(1);
    
    //Index of cell within domain
    const int ti = get_global_id(0) + 2; //Skip global ghost cells, i.e., +2
    const int tj = get_global_id(1) + 2;
    
    if (ti > 1 && ti < nx_+2 && tj > 1 && tj < ny_+2) {
        const int i = tx + 2; //Skip local ghost cells, i.e., +1
        const int j = ty + 2;
        
        Q[0][j][i] = Q[0][j][i] + (F[0][ty][tx] - F[0][ty][tx+1]) * dt_ / dx_;
        Q[1][j][i] = Q[1][j][i] + (F[1][ty][tx] - F[1][ty][tx+1]) * dt_ / dx_;
        Q[2][j][i] = Q[2][j][i] + (F[2][ty][tx] - F[2][ty][tx+1]) * dt_ / dx_;
    }
}






/**
  * Evolves the solution in time along the y axis (dimensional splitting)
  */
__device__ void evolveG1(float Q[3][block_height+2][block_width+2],
              float G[3][block_height+1][block_width+1],
              const int nx_, const int ny_,
              const float dy_, const float dt_) {
    //Index of thread within block
    const int tx = get_local_id(0);
    const int ty = get_local_id(1);
    
    //Index of cell within domain
    const int ti = get_global_id(0) + 1; //Skip global ghost cells, i.e., +1
    const int tj = get_global_id(1) + 1;
    
    if (ti > 0 && ti < nx_+1 && tj > 0 && tj < ny_+1) {
        const int i = tx + 1; //Skip local ghost cells, i.e., +1
        const int j = ty + 1;
        
        Q[0][j][i] = Q[0][j][i] + (G[0][ty][tx] - G[0][ty+1][tx]) * dt_ / dy_;
        Q[1][j][i] = Q[1][j][i] + (G[1][ty][tx] - G[1][ty+1][tx]) * dt_ / dy_;
        Q[2][j][i] = Q[2][j][i] + (G[2][ty][tx] - G[2][ty+1][tx]) * dt_ / dy_;
    }
}







/**
  * Evolves the solution in time along the y axis (dimensional splitting)
  */
__device__ void evolveG2(float Q[3][block_height+4][block_width+4],
              float G[3][block_height+1][block_width+1],
              const int nx_, const int ny_,
              const float dy_, const float dt_) {
    //Index of thread within block
    const int tx = get_local_id(0);
    const int ty = get_local_id(1);
    
    //Index of cell within domain
    const int ti = get_global_id(0) + 2; //Skip global ghost cells, i.e., +2
    const int tj = get_global_id(1) + 2;
    
    if (ti > 1 && ti < nx_+2 && tj > 1 && tj < ny_+2) {
        const int i = tx + 2; //Skip local ghost cells, i.e., +2
        const int j = ty + 2;
        
        Q[0][j][i] = Q[0][j][i] + (G[0][ty][tx] - G[0][ty+1][tx]) * dt_ / dy_;
        Q[1][j][i] = Q[1][j][i] + (G[1][ty][tx] - G[1][ty+1][tx]) * dt_ / dy_;
        Q[2][j][i] = Q[2][j][i] + (G[2][ty][tx] - G[2][ty+1][tx]) * dt_ / dy_;
    }
}










/**
  * Reconstructs a slope using the minmod limiter based on three 
  * consecutive values
  */
__device__ float minmodSlope(float left, float center, float right, float theta) {
    const float backward = (center - left) * theta;
    const float central = (right - left) * 0.5f;
    const float forward = (right - center) * theta;
    
	return 0.25f
		*copysign(1.0f, backward)
		*(copysign(1.0f, backward) + copysign(1.0f, central))
		*(copysign(1.0f, central) + copysign(1.0f, forward))
		*min( min(fabs(backward), fabs(central)), fabs(forward) );
}




/**
  * Reconstructs a minmod slope for a whole block along x
  */
__device__ void minmodSlopeX(float  Q[3][block_height+4][block_width+4],
                  float Qx[3][block_height+2][block_width+2],
                  const float theta_) {
    //Index of thread within block
    const int tx = get_local_id(0);
    const int ty = get_local_id(1);
    
    //Reconstruct slopes along x axis
    for (int j=ty; j<block_height; j+=get_local_size(1)) {
        const int l = j + 2; //Skip ghost cells
        for (int i=tx; i<block_width+2; i+=get_local_size(0)) {
            const int k = i + 1;
            for (int p=0; p<3; ++p) {
                Qx[p][j][i] = minmodSlope(Q[p][l][k-1], Q[p][l][k], Q[p][l][k+1], theta_);
            }
        }
    }
}


/**
  * Reconstructs a minmod slope for a whole block along y
  */
__device__ void minmodSlopeY(float  Q[3][block_height+4][block_width+4],
                  float Qy[3][block_height+2][block_width+2],
                  const float theta_) {
    //Index of thread within block
    const int tx = get_local_id(0);
    const int ty = get_local_id(1);
    
    for (int j=ty; j<block_height+2; j+=get_local_size(1)) {
        const int l = j + 1;
        for (int i=tx; i<block_width; i+=get_local_size(0)) {            
            const int k = i + 2; //Skip ghost cells
            for (int p=0; p<3; ++p) {
                Qy[p][j][i] = minmodSlope(Q[p][l-1][k], Q[p][l][k], Q[p][l+1][k], theta_);
            }
        }
    }
}









__device__ float3 F_func(const float3 Q, const float g) {
    float3 F;

    F.x = Q.y;                              //hu
    F.y = Q.y*Q.y / Q.x + 0.5f*g*Q.x*Q.x;   //hu*hu/h + 0.5f*g*h*h;
    F.z = Q.y*Q.z / Q.x;                    //hu*hv/h;

    return F;
}





/**
  * Central upwind flux function
  */
__device__ float3 CentralUpwindFlux(const float3 Qm, float3 Qp, const float g) {
    const float3 Fp = F_func(Qp, g);
    const float up = Qp.y / Qp.x;   // hu / h
    const float cp = sqrt(g*Qp.x); // sqrt(g*h)

    const float3 Fm = F_func(Qm, g);
    const float um = Qm.y / Qm.x;   // hu / h
    const float cm = sqrt(g*Qm.x); // sqrt(g*h)
    
    const float am = min(min(um-cm, up-cp), 0.0f); // largest negative wave speed
    const float ap = max(max(um+cm, up+cp), 0.0f); // largest positive wave speed
    
    return ((ap*Fm - am*Fp) + ap*am*(Qp-Qm))/(ap-am);
}










/**
  * Harten-Lax-van Leer with contact discontinuity (Toro 2001, p 180)
  */
__device__ float3 HLL_flux(const float3 Q_l, const float3 Q_r, const float g_) {    
    const float h_l = Q_l.x;
    const float h_r = Q_r.x;
    
    // Calculate velocities
    const float u_l = Q_l.y / h_l;
    const float u_r = Q_r.y / h_r;
    
    // Estimate the potential wave speeds
    const float c_l = sqrt(g_*h_l);
    const float c_r = sqrt(g_*h_r);
    
    // Compute h in the "star region", h^dagger
    const float h_dag = 0.5f * (h_l+h_r) - 0.25f * (u_r-u_l)*(h_l+h_r)/(c_l+c_r);
    
    const float q_l_tmp = sqrt(0.5f * ( (h_dag+h_l)*h_dag / (h_l*h_l) ) );
    const float q_r_tmp = sqrt(0.5f * ( (h_dag+h_r)*h_dag / (h_r*h_r) ) );
    
    const float q_l = (h_dag > h_l) ? q_l_tmp : 1.0f;
    const float q_r = (h_dag > h_r) ? q_r_tmp : 1.0f;
    
    // Compute wave speed estimates
    const float S_l = u_l - c_l*q_l;
    const float S_r = u_r + c_r*q_r;
    
    //Upwind selection
    if (S_l >= 0.0f) {
        return F_func(Q_l, g_);
    }
    else if (S_r <= 0.0f) {
        return F_func(Q_r, g_);
    }
    //Or estimate flux in the star region
    else {
        const float3 F_l = F_func(Q_l, g_);
        const float3 F_r = F_func(Q_r, g_);
        const float3 flux = (S_r*F_l - S_l*F_r + S_r*S_l*(Q_r - Q_l)) / (S_r-S_l);
        return flux;
    }
}









/**
  * Harten-Lax-van Leer with contact discontinuity (Toro 2001, p 181)
  */
__device__ float3 HLLC_flux(const float3 Q_l, const float3 Q_r, const float g_) {    
    const float h_l = Q_l.x;
    const float h_r = Q_r.x;
    
    // Calculate velocities
    const float u_l = Q_l.y / h_l;
    const float u_r = Q_r.y / h_r;
    
    // Estimate the potential wave speeds
    const float c_l = sqrt(g_*h_l);
    const float c_r = sqrt(g_*h_r);
    
    // Compute h in the "star region", h^dagger
    const float h_dag = 0.5f * (h_l+h_r) - 0.25f * (u_r-u_l)*(h_l+h_r)/(c_l+c_r);
    
    const float q_l_tmp = sqrt(0.5f * ( (h_dag+h_l)*h_dag / (h_l*h_l) ) );
    const float q_r_tmp = sqrt(0.5f * ( (h_dag+h_r)*h_dag / (h_r*h_r) ) );
    
    const float q_l = (h_dag > h_l) ? q_l_tmp : 1.0f;
    const float q_r = (h_dag > h_r) ? q_r_tmp : 1.0f;
    
    // Compute wave speed estimates
    const float S_l = u_l - c_l*q_l;
    const float S_r = u_r + c_r*q_r;
    const float S_star = ( S_l*h_r*(u_r - S_r) - S_r*h_l*(u_l - S_l) ) / ( h_r*(u_r - S_r) - h_l*(u_l - S_l) );
    
    const float3 F_l = F_func(Q_l, g_);
    const float3 F_r = F_func(Q_r, g_);
    
    //Upwind selection
    if (S_l >= 0.0f) {
        return F_l;
    }
    else if (S_r <= 0.0f) {
        return F_r;
    }
    //Or estimate flux in the "left star" region
    else if (S_l <= 0.0f && 0.0f <=S_star) {
        const float v_l = Q_l.z / h_l;
        const float3 Q_star_l = h_l * (S_l - u_l) / (S_l - S_star) * make_float3(1, S_star, v_l);
        const float3 flux = F_l + S_l*(Q_star_l - Q_l);
        return flux;
    }
    //Or estimate flux in the "righ star" region
    else if (S_star <= 0.0f && 0.0f <=S_r) {
        const float v_r = Q_r.z / h_r;
        const float3 Q_star_r = h_r * (S_r - u_r) / (S_r - S_star) * make_float3(1, S_star, v_r);
        const float3 flux = F_r + S_r*(Q_star_r - Q_r);
        return flux;
    }
    else {
        return make_float3(-99999.9f, -99999.9f, -99999.9f); //Something wrong here
    }
}



/**
  * Superbee flux limiter for WAF.
  * Related to superbee limiter so that WAF_superbee(r, c) = 1 - (1-|c|)*superbee(r)
  * @param r_ the ratio of upwind change (see Toro 2001, p. 203/204)
  * @param c_ the courant number for wave k, dt*S_k/dx
  */
__device__ float WAF_superbee(float r_, float c_) {
    // r <= 0.0
    if (r_ <= 0.0f) { 
        return 1.0f;
    }
    // 0.0 <= r <= 1/2
    else if (r_ <= 0.5f) { 
        return 1.0f - 2.0f*(1.0f - fabs(c_))*r_;
    }
    // 1/2 <= r <= 1
    else if (r_ <= 1.0f) {
        return fabs(c_);
    }
    // 1 <= r <= 2
    else  if (r_ <= 2.0f) {
        return 1.0f - (1.0f - fabs(c_))*r_;
    }
    // r >= 2
    else {
        return 2.0f*fabs(c_) - 1.0f;
    }
}




__device__ float WAF_albada(float r_, float c_) {
    if (r_ <= 0.0f) {
        return 1.0f;
    }
    else {
        return 1.0f - (1.0f - fabs(c_)) * r_ * (1.0f + r_) / (1.0f + r_*r_);
    }
}

__device__ float WAF_minmod(float r_, float c_) {
    return 1.0f - (1.0f - fabs(c_)) * fmax(0.0f, fmin(1.0f, r_));
}

__device__ float minmod(float r_) {
    return fmax(0.0f, fmin(1.0f, r_));
}

__device__ float superbee(float r_) {
    return fmax(0.0f, fmax(fmin(2.0f*r_, 1.0f), fmin(r_, 2.0f)));
}

__device__ float vanAlbada1(float r_) {
    return (r_*r_ + r_) / (r_*r_ + 1.0f);
}

__device__ float vanLeer(float r_) {
    return (r_ + fabs(r_)) / (1.0f + fabs(r_));
}

__device__ float limiterToWAFLimiter(float r_, float c_) {
    return 1.0f - (1.0f - fabs(c_))*r_;
}


/**
  * Weighted average flux (Toro 2001, p 200) for interface {i+1/2}
  * @param r_ The flux limiter parameter (see Toro 2001, p. 203)
  * @param Q_l2 Q_{i-1}
  * @param Q_l1 Q_{i}
  * @param Q_r1 Q_{i+1}
  * @param Q_r2 Q_{i+2}
  */
__device__ float3 WAF_1D_flux(const float3 Q_l2, const float3 Q_l1, const float3 Q_r1, const float3 Q_r2, const float g_, const float dx_, const float dt_) {     
    const float h_l = Q_l1.x;
    const float h_r = Q_r1.x;
    
    const float h_l2 = Q_l2.x;
    const float h_r2 = Q_r2.x;
    
    // Calculate velocities
    const float u_l = Q_l1.y / h_l;
    const float u_r = Q_r1.y / h_r;
    
    const float v_l = Q_l1.z / h_l;
    const float v_r = Q_r1.z / h_r;
    
    const float v_l2 = Q_l2.z / h_l2;
    const float v_r2 = Q_r2.z / h_r2;
    
    // Estimate the potential wave speeds
    const float c_l = sqrt(g_*h_l);
    const float c_r = sqrt(g_*h_r);
    
    // Compute h in the "star region", h^dagger
    const float h_dag = 0.5f * (h_l+h_r) - 0.25f * (u_r-u_l)*(h_l+h_r)/(c_l+c_r);
    
    const float q_l_tmp = sqrt(0.5f * ( (h_dag+h_l)*h_dag / (h_l*h_l) ) );
    const float q_r_tmp = sqrt(0.5f * ( (h_dag+h_r)*h_dag / (h_r*h_r) ) );
    
    const float q_l = (h_dag > h_l) ? q_l_tmp : 1.0f;
    const float q_r = (h_dag > h_r) ? q_r_tmp : 1.0f;
    
    // Compute wave speed estimates
    const float S_l = u_l - c_l*q_l; //FIXME: Right wave speed estimate?
    const float S_r = u_r + c_r*q_r;
    const float S_star = ( S_l*h_r*(u_r - S_r) - S_r*h_l*(u_l - S_l) ) / ( h_r*(u_r - S_r) - h_l*(u_l - S_l) );
    
    const float3 Q_star_l = h_l * (S_l - u_l) / (S_l - S_star) * make_float3(1, S_star, v_l);
    const float3 Q_star_r = h_r * (S_r - u_r) / (S_r - S_star) * make_float3(1, S_star, v_r);
    
    // Estimate the fluxes in the four regions
    const float3 F_1 = F_func(Q_l1, g_);
    const float3 F_4 = F_func(Q_r1, g_);
    
    const float3 F_2 = F_1 + S_l*(Q_star_l - Q_l1);
    const float3 F_3 = F_4 + S_r*(Q_star_r - Q_r1);
    //const float3 F_2 = F_func(Q_star_l, g_);
    //const float3 F_3 = F_func(Q_star_r, g_);
    
    // Compute the courant numbers for the waves
    const float c_1 = S_l * dt_ / dx_;
    const float c_2 = S_star * dt_ / dx_;
    const float c_3 = S_r * dt_ / dx_;
    
    // Compute the "upwind change" vectors for the i-3/2 and i+3/2 interfaces
    const float rh_m = (h_l - h_l2) / (h_r - h_l);
    const float rh_p = (h_r2 - h_r) / (h_r - h_l);
    
    const float rv_m = (v_l - v_l2) / (v_r - v_l);
    const float rv_p = (v_r2 - v_r) / (v_r - v_l);
    
    // Compute the r parameters for the flux limiter
    const float rh_1 = (c_1 > 0.0f) ? rh_m : rh_p; 
    const float rv_1 = (c_1 > 0.0f) ? rv_m : rv_p; 
    
    const float rh_2 = (c_2 > 0.0f) ? rh_m : rh_p; 
    const float rv_2 = (c_2 > 0.0f) ? rv_m : rv_p; 
    
    const float rh_3 = (c_3 > 0.0f) ? rh_m : rh_p;
    const float rv_3 = (c_3 > 0.0f) ? rv_m : rv_p;
    
    // Compute the limiter
    // We use h for the nonlinear waves, and v for the middle shear wave 
    ///**
    const float A_1 = copysign(1.0f, c_1) * WAF_minmod(rh_1, c_1);
    const float A_2 = copysign(1.0f, c_2) * WAF_minmod(rv_2, c_2); //Middle shear wave 
    const float A_3 = copysign(1.0f, c_3) * WAF_minmod(rh_3, c_3); 
    //*/
    /**
    //2nd order for smooth cases (unstable for shocks)
    const float A_1 = c_1;
    const float A_2 = c_2;
    const float A_3 = c_3;
    */
    /*
    const float A_1 = sign(c_1) * limiterToWAFLimiter(minmod(rh_1), c_1);
    const float A_2 = sign(c_2) * limiterToWAFLimiter(minmod(rv_2), c_2);
    const float A_3 = sign(c_3) * limiterToWAFLimiter(minmod(rh_3), c_3);
    */
        
    //Average the fluxes
    const float3 flux = 0.5f*( F_1 + F_4 )
                      - 0.5f*( A_1 * (F_2 - F_1)
                             + A_2 * (F_3 - F_2)
                             + A_3 * (F_4 - F_3) );

    /*
    const float d_0 = -1.0f;
    const float d_1 = -0.5f;//max(min(sign(c_1)*WAF_minbee(rh_1, c_1), 1.0f), -1.0f);
    const float d_2 = 0.0f;//max(min(sign(c_2)*WAF_minbee(rh_2, c_2), 1.0f), -1.0f);
    const float d_3 = 0.5f;//max(min(sign(c_3)*WAF_minbee(rh_3, c_3), 1.0f), -1.0f);
    const float d_4 = 1.0f;
    const float3 flux = 0.5f*(d_1 - d_0) * F_1
                        + 0.5f*(d_2 - d_1) * F_2
                        + 0.5f*(d_3 - d_2) * F_3
                        + 0.5f*(d_4 - d_3) * F_4;
    */
    /*
    const float3 F_hllc = (S_r*F_1 - S_l*F_4 + S_r*S_l*(Q_r1 - Q_l1)) / (S_r-S_l);
    const float3 flux = 0.5f*(d_1 - d_0) * F_1
                        + 0.5f*(d_3 - d_1) * F_hllc
                        + 0.5f*(d_4 - d_3) * F_4;
      */
                             /*
    const float c_0 = -1.0f;
    const float c_4 = 1.0f;
    const float3 flux = 0.5f*(c_1 - c_0) * F_1
                        + 0.5f*(c_2 - c_1) * F_2
                        + 0.5f*(c_3 - c_2) * F_3
                        + 0.5f*(c_4 - c_3) * F_4;
                        */
    //const float3 flux = 0.5f*( F_1 + F_4 ) - 0.5f*( sign(c_3) * A_3 * (F_4 - F_3) );
    return flux;
}







/**
  * Lax-Friedrichs flux (Toro 2001, p 163)
  */
__device__ float3 LxF_1D_flux(const float3 Q_l, const float3 Q_r, const float g_, const float dx_, const float dt_) {
    const float3 F_l = F_func(Q_l, g_);
    const float3 F_r = F_func(Q_r, g_);
    
    return 0.5f*(F_l + F_r) + (dx_/(2.0f*dt_))*(Q_l - Q_r);
}



/**
  * Lax-Friedrichs extended to 2D
  */
__device__ float3 LxF_2D_flux(const float3 Q_l, const float3 Q_r, const float g_, const float dx_, const float dt_) {
    const float3 F_l = F_func(Q_l, g_);
    const float3 F_r = F_func(Q_r, g_);
    
    //Note numerical diffusion for 2D here (0.25)
    return 0.5f*(F_l + F_r) + (dx_/(4.0f*dt_))*(Q_l - Q_r);
}




/**
  * Richtmeyer / Two-step Lax-Wendroff flux (Toro 2001, p 164)
  */
__device__ float3 LxW2_1D_flux(const float3 Q_l, const float3 Q_r, const float g_, const float dx_, const float dt_) {
    const float3 F_l = F_func(Q_l, g_);
    const float3 F_r = F_func(Q_r, g_);
    
    const float3 Q_lw2 = 0.5f*(Q_l + Q_r) + (dt_/(2.0f*dx_))*(F_l - F_r);
    
    return F_func(Q_lw2, g_);
}






/**
  * Godunovs centered scheme (Toro 2001, p 165)
  */
__device__ float3 GodC_1D_flux(const float3 Q_l, const float3 Q_r, const float g_, const float dx_, const float dt_) {
    const float3 F_l = F_func(Q_l, g_);
    const float3 F_r = F_func(Q_r, g_);
    
    const float3 Q_godc = 0.5f*(Q_l + Q_r) + (dt_/dx_)*(F_l - F_r);
    
    return F_func(Q_godc, g_);
}
    

    
    
/**
  * First Ordered Centered (Toro 2001, p.163)
  */
__device__ float3 FORCE_1D_flux(const float3 Q_l, const float3 Q_r, const float g_, const float dx_, const float dt_) {
    const float3 F_lf = LxF_1D_flux(Q_l, Q_r, g_, dx_, dt_);
    const float3 F_lw2 = LxW2_1D_flux(Q_l, Q_r, g_, dx_, dt_);
    return 0.5f*(F_lf + F_lw2);
}





