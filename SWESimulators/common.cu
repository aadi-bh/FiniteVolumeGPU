#include "hip/hip_runtime.h"
/*
This OpenCL kernel implements the Kurganov-Petrova numerical scheme 
for the shallow water equations, described in 
A. Kurganov & Guergana Petrova
A Second-Order Well-Balanced Positivity Preserving Central-Upwind
Scheme for the Saint-Venant System Communications in Mathematical
Sciences, 5 (2007), 133-160. 

Copyright (C) 2016  SINTEF ICT

This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/


/**
  * Location of thread in block
  */
inline __device__ int get_local_id(int dim) {
    switch(dim) {
        case 0: return threadIdx.x; 
        case 1: return threadIdx.y;
        case 2: return threadIdx.z;
        default: return -1;
    }
}


/**
  * Get block index
  */
__device__ int get_group_id(int dim) {
    switch(dim) {
        case 0: return blockIdx.x;
        case 1: return blockIdx.y;
        case 2: return blockIdx.z;
        default: return -1;
    }
}

/**
  * Location of thread in global domain
  */
__device__ int get_global_id(int dim) {
    switch(dim) {
        case 0: return blockDim.x*blockIdx.x + threadIdx.x;
        case 1: return blockDim.y*blockIdx.y + threadIdx.y;
        case 2: return blockDim.z*blockIdx.z + threadIdx.z;
        default: return -1;
    }
}


/**
  * Float3 operators 
  */
inline __device__ float3 operator*(const float a, const float3 b) {
    return make_float3(a*b.x, a*b.y, a*b.z);
}

inline __device__ float3 operator/(const float3 a, const float b) {
    return make_float3(a.x/b, a.y/b, a.z/b);
}

inline __device__ float3 operator-(const float3 a, const float3 b) {
    return make_float3(a.x-b.x, a.y-b.y, a.z-b.z);
}

inline __device__ float3 operator+(const float3 a, const float3 b) {
    return make_float3(a.x+b.x, a.y+b.y, a.z+b.z);
}

inline __device__ __host__ float clamp(const float f, const float a, const float b) {
    return fmaxf(a, fminf(f, b));
}





/**
  * Reads a block of data  with one ghost cell for the shallow water equations
  */
__device__ void readBlock1(float* h_ptr_, int h_pitch_,
                float* hu_ptr_, int hu_pitch_,
                float* hv_ptr_, int hv_pitch_,
                float Q[3][BLOCK_HEIGHT+2][BLOCK_WIDTH+2], 
                const int nx_, const int ny_) {
    //Index of thread within block
    const int tx = get_local_id(0);
    const int ty = get_local_id(1);
    
    //Index of block within domain
    const int bx = BLOCK_WIDTH * get_group_id(0);
    const int by = BLOCK_HEIGHT * get_group_id(1);
    
    //Read into shared memory
    for (int j=ty; j<BLOCK_HEIGHT+2; j+=BLOCK_HEIGHT) {
        const int l = clamp(by + j, 0, ny_+1); // Out of bounds
        
        //Compute the pointer to current row in the arrays
        float* const h_row  = (float*) ((char*) h_ptr_  + h_pitch_*l);
        float* const hu_row = (float*) ((char*) hu_ptr_ + hu_pitch_*l);
        float* const hv_row = (float*) ((char*) hv_ptr_ + hv_pitch_*l);
        
        for (int i=tx; i<BLOCK_WIDTH+2; i+=BLOCK_WIDTH) {
            const int k = clamp(bx + i, 0, nx_+1); // Out of bounds
            
            Q[0][j][i] = h_row[k];
            Q[1][j][i] = hu_row[k];
            Q[2][j][i] = hv_row[k];
        }
    }
}





/**
  * Reads a block of data  with two ghost cells for the shallow water equations
  */
__device__ void readBlock2(float* h_ptr_, int h_pitch_,
                float* hu_ptr_, int hu_pitch_,
                float* hv_ptr_, int hv_pitch_,
                float Q[3][BLOCK_HEIGHT+4][BLOCK_WIDTH+4], 
                const int nx_, const int ny_) {
    //Index of thread within block
    const int tx = get_local_id(0);
    const int ty = get_local_id(1);
    
    //Index of block within domain
    const int bx = BLOCK_WIDTH * get_group_id(0);
    const int by = BLOCK_HEIGHT * get_group_id(1);
    
    //Read into shared memory
    for (int j=ty; j<BLOCK_HEIGHT+4; j+=BLOCK_HEIGHT) {
        const int l = clamp(by + j, 0, ny_+3); // Out of bounds
        
        //Compute the pointer to current row in the arrays
        float* const h_row  = (float*) ((char*) h_ptr_  + h_pitch_*l);
        float* const hu_row = (float*) ((char*) hu_ptr_ + hu_pitch_*l);
        float* const hv_row = (float*) ((char*) hv_ptr_ + hv_pitch_*l);
        
        for (int i=tx; i<BLOCK_WIDTH+4; i+=BLOCK_WIDTH) {
            const int k = clamp(bx + i, 0, nx_+3); // Out of bounds
            
            Q[0][j][i] = h_row[k];
            Q[1][j][i] = hu_row[k];
            Q[2][j][i] = hv_row[k];
        }
    }
}




/**
  * Writes a block of data to global memory for the shallow water equations.
  */
__device__ void writeBlock1(float* h_ptr_, int h_pitch_,
                 float* hu_ptr_, int hu_pitch_,
                 float* hv_ptr_, int hv_pitch_,
                 float Q[3][BLOCK_HEIGHT+2][BLOCK_WIDTH+2],
                 const int nx_, const int ny_) {
    //Index of thread within block
    const int tx = get_local_id(0);
    const int ty = get_local_id(1);
    
    //Index of cell within domain
    const int ti = get_global_id(0) + 1; //Skip global ghost cells, i.e., +1
    const int tj = get_global_id(1) + 1;
    
    //Only write internal cells
    if (ti > 0 && ti < nx_+1 && tj > 0 && tj < ny_+1) {
        const int i = tx + 1; //Skip local ghost cells, i.e., +1
        const int j = ty + 1;

        float* const h_row  = (float*) ((char*) h_ptr_  + h_pitch_*tj);
        float* const hu_row = (float*) ((char*) hu_ptr_ + hu_pitch_*tj);
        float* const hv_row = (float*) ((char*) hv_ptr_ + hv_pitch_*tj);
        
        h_row[ti]  = Q[0][j][i];
        hu_row[ti] = Q[1][j][i];
        hv_row[ti] = Q[2][j][i];
    }
}





/**
  * Writes a block of data to global memory for the shallow water equations.
  */
__device__ void writeBlock2(float* h_ptr_, int h_pitch_,
                 float* hu_ptr_, int hu_pitch_,
                 float* hv_ptr_, int hv_pitch_,
                 float Q[3][BLOCK_HEIGHT+4][BLOCK_WIDTH+4], 
                 const int nx_, const int ny_) {
    //Index of thread within block
    const int tx = get_local_id(0);
    const int ty = get_local_id(1);
    
    //Index of cell within domain
    const int ti = get_global_id(0) + 2; //Skip global ghost cells, i.e., +2
    const int tj = get_global_id(1) + 2;
    
    //Only write internal cells
    if (ti > 1 && ti < nx_+2 && tj > 1 && tj < ny_+2) {
        const int i = tx + 2; //Skip local ghost cells, i.e., +2
        const int j = ty + 2;

        float* const h_row  = (float*) ((char*) h_ptr_ + h_pitch_*tj);
        float* const hu_row = (float*) ((char*) hu_ptr_ + hu_pitch_*tj);
        float* const hv_row = (float*) ((char*) hv_ptr_ + hv_pitch_*tj);
        
        h_row[ti]  = Q[0][j][i];
        hu_row[ti] = Q[1][j][i];
        hv_row[ti] = Q[2][j][i];
    }
}






/**
  * No flow boundary conditions for the shallow water equations
  * with one ghost cell in each direction
  */
__device__ void noFlowBoundary1(float Q[3][BLOCK_HEIGHT+2][BLOCK_WIDTH+2], const int nx_, const int ny_) {
    //Global index
    const int ti = get_global_id(0) + 1; //Skip global ghost cells, i.e., +1
    const int tj = get_global_id(1) + 1;
    
    //Block-local indices
    const int tx = get_local_id(0);
    const int ty = get_local_id(1);
    
    const int i = tx + 1; //Skip local ghost cells, i.e., +1
    const int j = ty + 1;
    
    //Fix boundary conditions
    if (ti == 1) {
        Q[0][j][i-1] =  Q[0][j][i];
        Q[1][j][i-1] = -Q[1][j][i];
        Q[2][j][i-1] =  Q[2][j][i];
    }
    if (ti == nx_) {
        Q[0][j][i+1] =  Q[0][j][i];
        Q[1][j][i+1] = -Q[1][j][i];
        Q[2][j][i+1] =  Q[2][j][i];
    }
    if (tj == 1) {
        Q[0][j-1][i] =  Q[0][j][i];
        Q[1][j-1][i] =  Q[1][j][i];
        Q[2][j-1][i] = -Q[2][j][i];
    }
    if (tj == ny_) {
        Q[0][j+1][i] =  Q[0][j][i];
        Q[1][j+1][i] =  Q[1][j][i];
        Q[2][j+1][i] = -Q[2][j][i];
    }
}






/**
  * No flow boundary conditions for the shallow water equations
  * with two ghost cells in each direction
  */
__device__ void noFlowBoundary2(float Q[3][BLOCK_HEIGHT+4][BLOCK_WIDTH+4], const int nx_, const int ny_) {
    //Global index
    const int ti = get_global_id(0) + 2; //Skip global ghost cells, i.e., +2
    const int tj = get_global_id(1) + 2;
    
    //Block-local indices
    const int tx = get_local_id(0);
    const int ty = get_local_id(1);
    
    const int i = tx + 2; //Skip local ghost cells, i.e., +2
    const int j = ty + 2;
    
    if (ti == 2) {
        Q[0][j][i-1] =  Q[0][j][i];
        Q[1][j][i-1] = -Q[1][j][i];
        Q[2][j][i-1] =  Q[2][j][i];
        
        Q[0][j][i-2] =  Q[0][j][i+1];
        Q[1][j][i-2] = -Q[1][j][i+1];
        Q[2][j][i-2] =  Q[2][j][i+1];
    }
    if (ti == nx_+1) {
        Q[0][j][i+1] =  Q[0][j][i];
        Q[1][j][i+1] = -Q[1][j][i];
        Q[2][j][i+1] =  Q[2][j][i];
        
        Q[0][j][i+2] =  Q[0][j][i-1];
        Q[1][j][i+2] = -Q[1][j][i-1];
        Q[2][j][i+2] =  Q[2][j][i-1];
    }
    if (tj == 2) {
        Q[0][j-1][i] =  Q[0][j][i];
        Q[1][j-1][i] =  Q[1][j][i];
        Q[2][j-1][i] = -Q[2][j][i];
        
        Q[0][j-2][i] =  Q[0][j+1][i];
        Q[1][j-2][i] =  Q[1][j+1][i];
        Q[2][j-2][i] = -Q[2][j+1][i];
    }
    if (tj == ny_+1) {
        Q[0][j+1][i] =  Q[0][j][i];
        Q[1][j+1][i] =  Q[1][j][i];
        Q[2][j+1][i] = -Q[2][j][i];
        
        Q[0][j+2][i] =  Q[0][j-1][i];
        Q[1][j+2][i] =  Q[1][j-1][i];
        Q[2][j+2][i] = -Q[2][j-1][i];
    }
}






/**
  * Evolves the solution in time along the x axis (dimensional splitting)
  */
__device__ void evolveF1(float Q[3][BLOCK_HEIGHT+2][BLOCK_WIDTH+2],
              float F[3][BLOCK_HEIGHT+1][BLOCK_WIDTH+1],
              const int nx_, const int ny_,
              const float dx_, const float dt_) {
    //Index of thread within block
    const int tx = get_local_id(0);
    const int ty = get_local_id(1);
    
    //Index of cell within domain
    const int ti = get_global_id(0) + 1; //Skip global ghost cells, i.e., +1
    const int tj = get_global_id(1) + 1;
    
    if (ti > 0 && ti < nx_+1 && tj > 0 && tj < ny_+1) {
        const int i = tx + 1; //Skip local ghost cells, i.e., +1
        const int j = ty + 1;
        
        Q[0][j][i] = Q[0][j][i] + (F[0][ty][tx] - F[0][ty][tx+1]) * dt_ / dx_;
        Q[1][j][i] = Q[1][j][i] + (F[1][ty][tx] - F[1][ty][tx+1]) * dt_ / dx_;
        Q[2][j][i] = Q[2][j][i] + (F[2][ty][tx] - F[2][ty][tx+1]) * dt_ / dx_;
    }
}






/**
  * Evolves the solution in time along the x axis (dimensional splitting)
  */
__device__ void evolveF2(float Q[3][BLOCK_HEIGHT+4][BLOCK_WIDTH+4],
              float F[3][BLOCK_HEIGHT+1][BLOCK_WIDTH+1],
              const int nx_, const int ny_,
              const float dx_, const float dt_) {
    //Index of thread within block
    const int tx = get_local_id(0);
    const int ty = get_local_id(1);
    
    //Index of cell within domain
    const int ti = get_global_id(0) + 2; //Skip global ghost cells, i.e., +2
    const int tj = get_global_id(1) + 2;
    
    if (ti > 1 && ti < nx_+2 && tj > 1 && tj < ny_+2) {
        const int i = tx + 2; //Skip local ghost cells, i.e., +1
        const int j = ty + 2;
        
        Q[0][j][i] = Q[0][j][i] + (F[0][ty][tx] - F[0][ty][tx+1]) * dt_ / dx_;
        Q[1][j][i] = Q[1][j][i] + (F[1][ty][tx] - F[1][ty][tx+1]) * dt_ / dx_;
        Q[2][j][i] = Q[2][j][i] + (F[2][ty][tx] - F[2][ty][tx+1]) * dt_ / dx_;
    }
}






/**
  * Evolves the solution in time along the y axis (dimensional splitting)
  */
__device__ void evolveG1(float Q[3][BLOCK_HEIGHT+2][BLOCK_WIDTH+2],
              float G[3][BLOCK_HEIGHT+1][BLOCK_WIDTH+1],
              const int nx_, const int ny_,
              const float dy_, const float dt_) {
    //Index of thread within block
    const int tx = get_local_id(0);
    const int ty = get_local_id(1);
    
    //Index of cell within domain
    const int ti = get_global_id(0) + 1; //Skip global ghost cells, i.e., +1
    const int tj = get_global_id(1) + 1;
    
    if (ti > 0 && ti < nx_+1 && tj > 0 && tj < ny_+1) {
        const int i = tx + 1; //Skip local ghost cells, i.e., +1
        const int j = ty + 1;
        
        Q[0][j][i] = Q[0][j][i] + (G[0][ty][tx] - G[0][ty+1][tx]) * dt_ / dy_;
        Q[1][j][i] = Q[1][j][i] + (G[1][ty][tx] - G[1][ty+1][tx]) * dt_ / dy_;
        Q[2][j][i] = Q[2][j][i] + (G[2][ty][tx] - G[2][ty+1][tx]) * dt_ / dy_;
    }
}







/**
  * Evolves the solution in time along the y axis (dimensional splitting)
  */
__device__ void evolveG2(float Q[3][BLOCK_HEIGHT+4][BLOCK_WIDTH+4],
              float G[3][BLOCK_HEIGHT+1][BLOCK_WIDTH+1],
              const int nx_, const int ny_,
              const float dy_, const float dt_) {
    //Index of thread within block
    const int tx = get_local_id(0);
    const int ty = get_local_id(1);
    
    //Index of cell within domain
    const int ti = get_global_id(0) + 2; //Skip global ghost cells, i.e., +2
    const int tj = get_global_id(1) + 2;
    
    if (ti > 1 && ti < nx_+2 && tj > 1 && tj < ny_+2) {
        const int i = tx + 2; //Skip local ghost cells, i.e., +2
        const int j = ty + 2;
        
        Q[0][j][i] = Q[0][j][i] + (G[0][ty][tx] - G[0][ty+1][tx]) * dt_ / dy_;
        Q[1][j][i] = Q[1][j][i] + (G[1][ty][tx] - G[1][ty+1][tx]) * dt_ / dy_;
        Q[2][j][i] = Q[2][j][i] + (G[2][ty][tx] - G[2][ty+1][tx]) * dt_ / dy_;
    }
}










__device__ float3 F_func(const float3 Q, const float g) {
    float3 F;

    F.x = Q.y;                              //hu
    F.y = Q.y*Q.y / Q.x + 0.5f*g*Q.x*Q.x;   //hu*hu/h + 0.5f*g*h*h;
    F.z = Q.y*Q.z / Q.x;                    //hu*hv/h;

    return F;
}

























