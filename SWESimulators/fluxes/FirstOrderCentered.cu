#include "hip/hip_runtime.h"
/*
This file implements the First ORder CEntered flux

Copyright (C) 2016, 2017, 2018 SINTEF ICT

This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

#include "./LaxFriedrichs.cu"
#include "./LaxWendroff.cu"


    
    
/**
  * First Ordered Centered (Toro 2001, p.163)
  */
__device__ float3 FORCE_1D_flux(const float3 Q_l, const float3 Q_r, const float g_, const float dx_, const float dt_) {
    const float3 F_lf = LxF_1D_flux(Q_l, Q_r, g_, dx_, dt_);
    const float3 F_lw2 = LxW2_1D_flux(Q_l, Q_r, g_, dx_, dt_);
    return 0.5f*(F_lf + F_lw2);
}
