#include "hip/hip_runtime.h"
/*
This file implements the Harten-Lax-van Leer flux

Copyright (C) 2016, 2017, 2018 SINTEF ICT

This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/


/**
  * Harten-Lax-van Leer with contact discontinuity (Toro 2001, p 180)
  */
__device__ float3 HLL_flux(const float3 Q_l, const float3 Q_r, const float g_) {    
    const float h_l = Q_l.x;
    const float h_r = Q_r.x;
    
    // Calculate velocities
    const float u_l = Q_l.y / h_l;
    const float u_r = Q_r.y / h_r;
    
    // Estimate the potential wave speeds
    const float c_l = sqrt(g_*h_l);
    const float c_r = sqrt(g_*h_r);
    
    // Compute h in the "star region", h^dagger
    const float h_dag = 0.5f * (h_l+h_r) - 0.25f * (u_r-u_l)*(h_l+h_r)/(c_l+c_r);
    
    const float q_l_tmp = sqrt(0.5f * ( (h_dag+h_l)*h_dag / (h_l*h_l) ) );
    const float q_r_tmp = sqrt(0.5f * ( (h_dag+h_r)*h_dag / (h_r*h_r) ) );
    
    const float q_l = (h_dag > h_l) ? q_l_tmp : 1.0f;
    const float q_r = (h_dag > h_r) ? q_r_tmp : 1.0f;
    
    // Compute wave speed estimates
    const float S_l = u_l - c_l*q_l;
    const float S_r = u_r + c_r*q_r;
    
    //Upwind selection
    if (S_l >= 0.0f) {
        return F_func(Q_l, g_);
    }
    else if (S_r <= 0.0f) {
        return F_func(Q_r, g_);
    }
    //Or estimate flux in the star region
    else {
        const float3 F_l = F_func(Q_l, g_);
        const float3 F_r = F_func(Q_r, g_);
        const float3 flux = (S_r*F_l - S_l*F_r + S_r*S_l*(Q_r - Q_l)) / (S_r-S_l);
        return flux;
    }
}
