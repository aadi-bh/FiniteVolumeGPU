#include "hip/hip_runtime.h"
/*
This file implements the Weighted Average Flux

Copyright (C) 2016, 2017, 2018 SINTEF ICT

This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

#include "limiters.cu"

/**
  * Superbee flux limiter for WAF.
  * Related to superbee limiter so that WAF_superbee(r, c) = 1 - (1-|c|)*superbee(r)
  * @param r_ the ratio of upwind change (see Toro 2001, p. 203/204)
  * @param c_ the courant number for wave k, dt*S_k/dx
  */
__device__ float WAF_superbee(float r_, float c_) {
    // r <= 0.0
    if (r_ <= 0.0f) { 
        return 1.0f;
    }
    // 0.0 <= r <= 1/2
    else if (r_ <= 0.5f) { 
        return 1.0f - 2.0f*(1.0f - fabsf(c_))*r_;
    }
    // 1/2 <= r <= 1
    else if (r_ <= 1.0f) {
        return fabs(c_);
    }
    // 1 <= r <= 2
    else  if (r_ <= 2.0f) {
        return 1.0f - (1.0f - fabsf(c_))*r_;
    }
    // r >= 2
    else {
        return 2.0f*fabsf(c_) - 1.0f;
    }
}




__device__ float WAF_albada(float r_, float c_) {
    if (r_ <= 0.0f) {
        return 1.0f;
    }
    else {
        return 1.0f - (1.0f - fabsf(c_)) * r_ * (1.0f + r_) / (1.0f + r_*r_);
    }
}

__device__ float WAF_minbee(float r_, float c_) {
    r_ = fmaxf(-1.0f, fminf(2.0f, r_));
    if (r_ <= 0.0f) {
        return 1.0f;
    }
    if (r_ >= 0.0f && r_ <= 1.0f) {
        return 1.0f - (1.0f - fabsf(c_)) * r_;
    }
    else {
        return fabsf(c_);
    }
}

__device__ float WAF_minmod(float r_, float c_) {
    return 1.0f - (1.0f - fabsf(c_)) * fmaxf(0.0f, fminf(1.0f, r_));
}



__device__ float limiterToWAFLimiter(float r_, float c_) {
    return 1.0f - (1.0f - fabsf(c_))*r_;
}

__device__ float desingularize(float x_, float eps_) {
    return copysign(1.0f, x_)*fmaxf(fabsf(x_), fminf(x_*x_/(2.0f*eps_)+0.5f*eps_, eps_));
}

// Compute h in the "star region", h^dagger
__device__ __inline__ float computeHStar(float h_l, float h_r, float u_l, float u_r, float c_l, float c_r, float g_) {
    
    //This estimate for the h* gives rise to spurious oscillations. 
    //return 0.5f * (h_l+h_r) - 0.25f * (u_r-u_l)*(h_l+h_r)/(c_l+c_r);
    
    const float h_tmp = 0.5f * (c_l + c_r) + 0.25f * (u_l - u_r);
    return h_tmp*h_tmp / g_;
}

/**
  * Weighted average flux (Toro 2001, p 200) for interface {i+1/2}
  * @param r_ The flux limiter parameter (see Toro 2001, p. 203)
  * @param Q_l2 Q_{i-1}
  * @param Q_l1 Q_{i}
  * @param Q_r1 Q_{i+1}
  * @param Q_r2 Q_{i+2}
  */
__device__ float3 WAF_1D_flux(const float3 Q_l2, const float3 Q_l1, const float3 Q_r1, const float3 Q_r2, const float g_, const float dx_, const float dt_) {     
    const float h_l = Q_l1.x;
    const float h_r = Q_r1.x;
    
    const float h_l2 = Q_l2.x;
    const float h_r2 = Q_r2.x;
    
    // Calculate velocities
    const float u_l = Q_l1.y / h_l;
    const float u_r = Q_r1.y / h_r;
    
    const float u_l2 = Q_l2.y / h_l2;
    const float u_r2 = Q_r2.y / h_r2;
    
    const float v_l = Q_l1.z / h_l;
    const float v_r = Q_r1.z / h_r;
    
    const float v_l2 = Q_l2.z / h_l2;
    const float v_r2 = Q_r2.z / h_r2;
    
    // Estimate the potential wave speeds
    const float c_l = sqrt(g_*h_l);
    const float c_r = sqrt(g_*h_r);
    
    const float c_l2 = sqrt(g_*h_l2);
    const float c_r2 = sqrt(g_*h_r2);
    
    // Compute h in the "star region", h^dagger
    const float h_dag_l = computeHStar(h_l2,  h_l, u_l2,  u_l, c_l2,  c_l, g_);
    const float h_dag   = computeHStar( h_l,  h_r,  u_l,  u_r,  c_l,  c_r, g_);
    const float h_dag_r = computeHStar( h_r, h_r2,  u_r, u_r2,  c_r, c_r2, g_);
    
    const float q_l_tmp = sqrt(0.5f * ( (h_dag+h_l)*h_dag ) ) / h_l;
    const float q_r_tmp = sqrt(0.5f * ( (h_dag+h_r)*h_dag ) ) / h_r;
    
    const float q_l = (h_dag > h_l) ? q_l_tmp : 1.0f;
    const float q_r = (h_dag > h_r) ? q_r_tmp : 1.0f;
    
    // Compute wave speed estimates
    const float S_l = u_l - c_l*q_l; 
    const float S_r = u_r + c_r*q_r;
    const float S_star = ( S_l*h_r*(u_r - S_r) - S_r*h_l*(u_l - S_l) ) / ( h_r*(u_r - S_r) - h_l*(u_l - S_l) );
    
    const float3 Q_star_l = h_l * (S_l - u_l) / (S_l - S_star) * make_float3(1.0, S_star, v_l);
    const float3 Q_star_r = h_r * (S_r - u_r) / (S_r - S_star) * make_float3(1.0, S_star, v_r);
    
    // Estimate the fluxes in the four regions
    const float3 F_1 = F_func(Q_l1, g_);
    const float3 F_4 = F_func(Q_r1, g_);
    
    const float3 F_2 = F_1 + S_l*(Q_star_l - Q_l1);
    const float3 F_3 = F_4 + S_r*(Q_star_r - Q_r1);
    //const float3 F_2 = F_func(Q_star_l, g_);
    //const float3 F_3 = F_func(Q_star_r, g_);
    
    // Compute the courant numbers for the waves
    const float c_1 = S_l * dt_ / dx_;
    const float c_2 = S_star * dt_ / dx_;
    const float c_3 = S_r * dt_ / dx_;
    
    // Compute the "upwind change" vectors for the i-3/2 and i+3/2 interfaces
    const float eps = 1.0e-6f;
    const float r_1 = desingularize( (c_1 > 0.0f) ? (h_dag_l - h_l2) : (h_dag_r - h_r), eps) / desingularize((h_dag - h_l), eps);
    const float r_2 = desingularize( (c_2 > 0.0f) ? (v_l - v_l2) : (v_r2 - v_r), eps ) / desingularize((v_r - v_l), eps);
    const float r_3 = desingularize( (c_3 > 0.0f) ? (h_l - h_dag_l) : (h_r2 - h_dag_r), eps ) / desingularize((h_r - h_dag), eps);
        
    // Compute the limiter
    // We use h for the nonlinear waves, and v for the middle shear wave 
    const float A_1 = copysign(1.0f, c_1) * limiterToWAFLimiter(generalized_minmod(r_1, 1.9f), c_1);
    const float A_2 = copysign(1.0f, c_2) * limiterToWAFLimiter(generalized_minmod(r_2, 1.9f), c_2); 
    const float A_3 = copysign(1.0f, c_3) * limiterToWAFLimiter(generalized_minmod(r_3, 1.9f), c_3);
    
    //Average the fluxes
    const float3 flux = 0.5f*( F_1 + F_4 )
                      - 0.5f*( A_1 * (F_2 - F_1)
                             + A_2 * (F_3 - F_2)
                             + A_3 * (F_4 - F_3) );

    return flux;
}
