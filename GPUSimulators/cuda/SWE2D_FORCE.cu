#include "hip/hip_runtime.h"
/*
This OpenCL kernel implements the classical Lax-Friedrichs scheme
for the shallow water equations, with edge fluxes.

Copyright (C) 2016  SINTEF ICT

This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/


#include "common.h"
#include "SWECommon.h"


/**
  * Computes the flux along the x axis for all faces
  */
__device__ 
void computeFluxF(float Q[3][BLOCK_HEIGHT+2][BLOCK_WIDTH+2],
                  float F[3][BLOCK_HEIGHT+1][BLOCK_WIDTH+1],
                  const float g_, const float dx_, const float dt_) {
                      
    //Index of thread within block
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    
    //Compute fluxes along the x axis
    {
        int j=ty;
        const int l = j + 1; //Skip ghost cells
        for (int i=tx; i<BLOCK_WIDTH+1; i+=BLOCK_WIDTH) {
            const int k = i;
            
            // Q at interface from the right and left
            const float3 Qp = make_float3(Q[0][l][k+1],
                                          Q[1][l][k+1],
                                          Q[2][l][k+1]);
            const float3 Qm = make_float3(Q[0][l][k],
                                          Q[1][l][k],
                                          Q[2][l][k]);
                                       
            // Computed flux
            const float3 flux = FORCE_1D_flux(Qm, Qp, g_, dx_, dt_);
            F[0][j][i] = flux.x;
            F[1][j][i] = flux.y;
            F[2][j][i] = flux.z;
        }
    }
}


/**
  * Computes the flux along the y axis for all faces
  */
__device__ 
void computeFluxG(float Q[3][BLOCK_HEIGHT+2][BLOCK_WIDTH+2],
                  float G[3][BLOCK_HEIGHT+1][BLOCK_WIDTH+1],
                  const float g_, const float dy_, const float dt_) {
    //Index of thread within block
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    
    //Compute fluxes along the y axis
    for (int j=ty; j<BLOCK_HEIGHT+1; j+=BLOCK_HEIGHT) {
        const int l = j;
        {
            int i=tx;
            const int k = i + 1; //Skip ghost cells
            
            // Q at interface from the right and left
            // Note that we swap hu and hv
            const float3 Qp = make_float3(Q[0][l+1][k],
                                          Q[2][l+1][k],
                                          Q[1][l+1][k]);
            const float3 Qm = make_float3(Q[0][l][k],
                                          Q[2][l][k],
                                          Q[1][l][k]);

            // Computed flux
            // Note that we swap back
            const float3 flux = FORCE_1D_flux(Qm, Qp, g_, dy_, dt_);
            G[0][j][i] = flux.x;
            G[1][j][i] = flux.z;
            G[2][j][i] = flux.y;
        }
    }
}


extern "C" {
__global__ void FORCEKernel(
        int nx_, int ny_,
        float dx_, float dy_, float dt_,
        float g_,
        
        //Input h^n
        float* h0_ptr_, int h0_pitch_,
        float* hu0_ptr_, int hu0_pitch_,
        float* hv0_ptr_, int hv0_pitch_,
        
        //Output h^{n+1}
        float* h1_ptr_, int h1_pitch_,
        float* hu1_ptr_, int hu1_pitch_,
        float* hv1_ptr_, int hv1_pitch_) {
    
    const unsigned int w = BLOCK_WIDTH;
    const unsigned int h = BLOCK_HEIGHT;
    const unsigned int gc = 1;
    
    __shared__ float Q[3][h+2][w+2];
    __shared__ float F[3][h+1][w+1];
    
    //Read into shared memory
    readBlock<w, h, gc>( h0_ptr_,  h0_pitch_, Q[0], nx_+2, ny_+2);
    readBlock<w, h, gc>(hu0_ptr_, hu0_pitch_, Q[1], nx_+2, ny_+2);
    readBlock<w, h, gc>(hv0_ptr_, hv0_pitch_, Q[2], nx_+2, ny_+2);
    __syncthreads();
    
    //Set boundary conditions
    noFlowBoundary<w, h, gc,  1,  1>(Q[0], nx_, ny_);
    noFlowBoundary<w, h, gc, -1,  1>(Q[1], nx_, ny_);
    noFlowBoundary<w, h, gc,  1, -1>(Q[2], nx_, ny_);
    __syncthreads();
    
    //Compute flux along x, and evolve
    computeFluxF(Q, F, g_, dx_, dt_);
    __syncthreads();
    
    evolveF<w, h, gc>(Q[0], F[0], dx_, dt_);
    evolveF<w, h, gc>(Q[1], F[1], dx_, dt_);
    evolveF<w, h, gc>(Q[2], F[2], dx_, dt_);
    __syncthreads();
    
    //Set boundary conditions
    noFlowBoundary<w, h, gc,  1,  1>(Q[0], nx_, ny_);
    noFlowBoundary<w, h, gc, -1,  1>(Q[1], nx_, ny_);
    noFlowBoundary<w, h, gc,  1, -1>(Q[2], nx_, ny_);
    __syncthreads();
    
    //Compute flux along y, and evolve
    computeFluxG(Q, F, g_, dy_, dt_);
    __syncthreads();
    
    evolveG<w, h, gc>(Q[0], F[0], dy_, dt_);
    evolveG<w, h, gc>(Q[1], F[1], dy_, dt_);
    evolveG<w, h, gc>(Q[2], F[2], dy_, dt_);
    __syncthreads();
    
    //Write to main memory
    writeBlock<w, h, gc>( h1_ptr_,  h1_pitch_, Q[0], nx_, ny_);
    writeBlock<w, h, gc>(hu1_ptr_, hu1_pitch_, Q[1], nx_, ny_);
    writeBlock<w, h, gc>(hv1_ptr_, hv1_pitch_, Q[2], nx_, ny_);
}

} // extern "C"