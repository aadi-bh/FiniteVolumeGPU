#include "hip/hip_runtime.h"
/*
This OpenCL kernel implements the Kurganov-Petrova numerical scheme 
for the shallow water equations, described in 
A. Kurganov & Guergana Petrova
A Second-Order Well-Balanced Positivity Preserving Central-Upwind
Scheme for the Saint-Venant System Communications in Mathematical
Sciences, 5 (2007), 133-160. 

Copyright (C) 2016  SINTEF ICT

This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/



#include "common.h"
#include "SWECommon.h"
#include "limiters.h"


__device__
void computeFluxF(float Q[3][BLOCK_HEIGHT+4][BLOCK_WIDTH+4],
                  float Qx[3][BLOCK_HEIGHT+2][BLOCK_WIDTH+2],
                  float F[3][BLOCK_HEIGHT+1][BLOCK_WIDTH+1],
                  const float g_) {
    //Index of thread within block
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    
    {
        int j=ty;
        const int l = j + 2; //Skip ghost cells
        for (int i=tx; i<BLOCK_WIDTH+1; i+=BLOCK_WIDTH) {
            const int k = i + 1;
            // Q at interface from the right and left
            const float3 Qp = make_float3(Q[0][l][k+1] - 0.5f*Qx[0][j][i+1],
                                          Q[1][l][k+1] - 0.5f*Qx[1][j][i+1],
                                          Q[2][l][k+1] - 0.5f*Qx[2][j][i+1]);
            const float3 Qm = make_float3(Q[0][l][k  ] + 0.5f*Qx[0][j][i  ],
                                          Q[1][l][k  ] + 0.5f*Qx[1][j][i  ],
                                          Q[2][l][k  ] + 0.5f*Qx[2][j][i  ]);
                                       
            // Computed flux
            const float3 flux = CentralUpwindFlux(Qm, Qp, g_);
            F[0][j][i] = flux.x;
            F[1][j][i] = flux.y;
            F[2][j][i] = flux.z;
        }
    }    
}

__device__
void computeFluxG(float Q[3][BLOCK_HEIGHT+4][BLOCK_WIDTH+4],
                  float Qy[3][BLOCK_HEIGHT+2][BLOCK_WIDTH+2],
                  float G[3][BLOCK_HEIGHT+1][BLOCK_WIDTH+1],
                  const float g_) {
    //Index of thread within block
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    
    for (int j=ty; j<BLOCK_HEIGHT+1; j+=BLOCK_HEIGHT) {
        const int l = j + 1;
        {
            int i=tx;
            const int k = i + 2; //Skip ghost cells
            // Q at interface from the right and left
            // Note that we swap hu and hv
            const float3 Qp = make_float3(Q[0][l+1][k] - 0.5f*Qy[0][j+1][i],
                                          Q[2][l+1][k] - 0.5f*Qy[2][j+1][i],
                                          Q[1][l+1][k] - 0.5f*Qy[1][j+1][i]);
            const float3 Qm = make_float3(Q[0][l  ][k] + 0.5f*Qy[0][j  ][i],
                                          Q[2][l  ][k] + 0.5f*Qy[2][j  ][i],
                                          Q[1][l  ][k] + 0.5f*Qy[1][j  ][i]);
                                       
            // Computed flux
            // Note that we swap back
            const float3 flux = CentralUpwindFlux(Qm, Qp, g_);
            G[0][j][i] = flux.x;
            G[1][j][i] = flux.z;
            G[2][j][i] = flux.y;
        }
    }
}




/**
  * This unsplit kernel computes the 2D numerical scheme with a TVD RK2 time integration scheme
  */
extern "C" {
__global__ void KP07Kernel(
        int nx_, int ny_,
        float dx_, float dy_, float dt_,
        float g_,
        
        float theta_,
        
        int step_,
        
        //Input h^n
        float* h0_ptr_, int h0_pitch_,
        float* hu0_ptr_, int hu0_pitch_,
        float* hv0_ptr_, int hv0_pitch_,
        
        //Output h^{n+1}
        float* h1_ptr_, int h1_pitch_,
        float* hu1_ptr_, int hu1_pitch_,
        float* hv1_ptr_, int hv1_pitch_) {
            
    const unsigned int w = BLOCK_WIDTH;
    const unsigned int h = BLOCK_HEIGHT;
    const unsigned int gc = 2;
    const unsigned int vars = 3;
        
    //Index of thread within block
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;

    //Index of cell within domain
    const int ti = blockDim.x*blockIdx.x + threadIdx.x + 2; //Skip global ghost cells, i.e., +2
    const int tj = blockDim.y*blockIdx.y + threadIdx.y + 2;
    
    //Shared memory variables
    __shared__ float Q[3][h+4][w+4];
    
    //The following slightly wastes memory, but enables us to reuse the 
    //funcitons in common.opencl
    __shared__ float Qx[3][h+2][w+2];
    __shared__ float Qy[3][h+2][w+2];
    __shared__ float  F[3][h+1][w+1];
    __shared__ float  G[3][h+1][w+1];
    
    
    
    //Read into shared memory
    readBlock<w, h, gc>( h0_ptr_,  h0_pitch_, Q[0], nx_+2, ny_+2);
    readBlock<w, h, gc>(hu0_ptr_, hu0_pitch_, Q[1], nx_+2, ny_+2);
    readBlock<w, h, gc>(hv0_ptr_, hv0_pitch_, Q[2], nx_+2, ny_+2);
    __syncthreads();
    
    
    //Fix boundary conditions
    noFlowBoundary<w, h, gc,  1,  1>(Q[0], nx_, ny_);
    noFlowBoundary<w, h, gc, -1,  1>(Q[1], nx_, ny_);
    noFlowBoundary<w, h, gc,  1, -1>(Q[2], nx_, ny_);
    __syncthreads();
    
    
    //Reconstruct slopes along x and axis
    minmodSlopeX(Q, Qx, theta_);
    minmodSlopeY(Q, Qy, theta_);
    __syncthreads();
    
    
    //Compute fluxes along the x and y axis
    computeFluxF(Q, Qx, F, g_);
    computeFluxG(Q, Qy, G, g_);
    __syncthreads();
    
    
    //Sum fluxes and advance in time for all internal cells
    if (ti > 1 && ti < nx_+2 && tj > 1 && tj < ny_+2) {
        const int i = tx + 2; //Skip local ghost cells, i.e., +2
        const int j = ty + 2;
        
        const float h1  = Q[0][j][i] + (F[0][ty][tx] - F[0][ty  ][tx+1]) * dt_ / dx_ 
                                     + (G[0][ty][tx] - G[0][ty+1][tx  ]) * dt_ / dy_;
        const float hu1 = Q[1][j][i] + (F[1][ty][tx] - F[1][ty  ][tx+1]) * dt_ / dx_ 
                                     + (G[1][ty][tx] - G[1][ty+1][tx  ]) * dt_ / dy_;
        const float hv1 = Q[2][j][i] + (F[2][ty][tx] - F[2][ty  ][tx+1]) * dt_ / dx_ 
                                     + (G[2][ty][tx] - G[2][ty+1][tx  ]) * dt_ / dy_;

        float* const h_row  = (float*) ((char*) h1_ptr_ + h1_pitch_*tj);
        float* const hu_row = (float*) ((char*) hu1_ptr_ + hu1_pitch_*tj);
        float* const hv_row = (float*) ((char*) hv1_ptr_ + hv1_pitch_*tj);
                    
        if  (step_ == 0) {
            //First step of RK2 ODE integrator
            
            h_row[ti] = h1;
            hu_row[ti] = hu1;
            hv_row[ti] = hv1;
        }
        else if (step_ == 1) {
            //Second step of RK2 ODE integrator
            
            //First read Q^n
            const float h_a  = h_row[ti];
            const float hu_a = hu_row[ti];
            const float hv_a = hv_row[ti];
            
            //Compute Q^n+1
            const float h_b  = 0.5f*(h_a + h1);
            const float hu_b = 0.5f*(hu_a + hu1);
            const float hv_b = 0.5f*(hv_a + hv1);
            
            //Write to main memory
            h_row[ti] = h_b;
            hu_row[ti] = hu_b;
            hv_row[ti] = hv_b;
        }
    }
}
} //extern "C"