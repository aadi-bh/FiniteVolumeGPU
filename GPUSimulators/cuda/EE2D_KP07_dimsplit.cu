#include "hip/hip_runtime.h"
 /*
This kernel implements the Central Upwind flux function to
solve the Euler equations 

Copyright (C) 2018  SINTEF Digital

This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/



#include "common.h"
#include "EulerCommon.h"
#include "limiters.h"


__device__
void computeFluxF(float Q[4][BLOCK_HEIGHT+4][BLOCK_WIDTH+4],
                  float Qx[4][BLOCK_HEIGHT+2][BLOCK_WIDTH+2],
                  float F[4][BLOCK_HEIGHT+1][BLOCK_WIDTH+1],
                  const float gamma_, const float dx_, const float dt_) {
    //Index of thread within block
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    
    {
        int j=ty;
        const int l = j + 2; //Skip ghost cells
        for (int i=tx; i<BLOCK_WIDTH+1; i+=BLOCK_WIDTH) {
            const int k = i + 1;
            // Reconstruct point values of Q at the left and right hand side 
            // of the cell for both the left (i) and right (i+1) cell 
            const float4 Q_rl = make_float4(Q[0][l][k+1] - 0.5f*Qx[0][j][i+1],
                                            Q[1][l][k+1] - 0.5f*Qx[1][j][i+1],
                                            Q[2][l][k+1] - 0.5f*Qx[2][j][i+1],
                                            Q[4][l][k+1] - 0.5f*Qx[4][j][i+1]);
            const float4 Q_rr = make_float4(Q[0][l][k+1] + 0.5f*Qx[0][j][i+1],
                                            Q[1][l][k+1] + 0.5f*Qx[1][j][i+1],
                                            Q[2][l][k+1] + 0.5f*Qx[2][j][i+1],
                                            Q[4][l][k+1] + 0.5f*Qx[4][j][i+1]);
                                         
            const float4 Q_ll = make_float4(Q[0][l][k] - 0.5f*Qx[0][j][i],
                                            Q[1][l][k] - 0.5f*Qx[1][j][i],
                                            Q[2][l][k] - 0.5f*Qx[2][j][i],
                                            Q[4][l][k] - 0.5f*Qx[4][j][i]);
            const float4 Q_lr = make_float4(Q[0][l][k] + 0.5f*Qx[0][j][i],
                                            Q[1][l][k] + 0.5f*Qx[1][j][i],
                                            Q[2][l][k] + 0.5f*Qx[2][j][i],
                                            Q[4][l][k] + 0.5f*Qx[4][j][i]);
                                    
            //Evolve half a timestep (predictor step)
            const float4 Q_r_bar = Q_rl + dt_/(2.0f*dx_) * (F_func(Q_rl, gamma_) - F_func(Q_rr, gamma_));
            const float4 Q_l_bar = Q_lr + dt_/(2.0f*dx_) * (F_func(Q_ll, gamma_) - F_func(Q_lr, gamma_));

            // Compute flux based on prediction
            const float4 flux = CentralUpwindFlux(Q_l_bar, Q_r_bar, gamma_);
            
            //Write to shared memory
            F[0][j][i] = flux.x;
            F[1][j][i] = flux.y;
            F[2][j][i] = flux.z;
            F[3][j][i] = flux.w;
        }
    }    
}

__device__
void computeFluxG(float Q[4][BLOCK_HEIGHT+4][BLOCK_WIDTH+4],
                  float Qy[4][BLOCK_HEIGHT+2][BLOCK_WIDTH+2],
                  float G[4][BLOCK_HEIGHT+1][BLOCK_WIDTH+1],
                  const float gamma_, const float dy_, const float dt_) {
    //Index of thread within block
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    
    for (int j=ty; j<BLOCK_HEIGHT+1; j+=BLOCK_HEIGHT) {
        const int l = j + 1;
        {
            int i=tx;
            const int k = i + 2; //Skip ghost cells
            // Reconstruct point values of Q at the left and right hand side 
            // of the cell for both the left (i) and right (i+1) cell 
            //NOte that hu and hv are swapped ("transposing" the domain)!
            const float4 Q_rl = make_float4(Q[0][l+1][k] - 0.5f*Qy[0][j+1][i],
                                            Q[2][l+1][k] - 0.5f*Qy[2][j+1][i],
                                            Q[1][l+1][k] - 0.5f*Qy[1][j+1][i],
                                            Q[3][l+1][k] - 0.5f*Qy[3][j+1][i]);
            const float4 Q_rr = make_float4(Q[0][l+1][k] + 0.5f*Qy[0][j+1][i],
                                            Q[2][l+1][k] + 0.5f*Qy[2][j+1][i],
                                            Q[1][l+1][k] + 0.5f*Qy[1][j+1][i],
                                            Q[3][l+1][k] + 0.5f*Qy[3][j+1][i]);
                                       
            const float4 Q_ll = make_float4(Q[0][l][k] - 0.5f*Qy[0][j][i],
                                            Q[2][l][k] - 0.5f*Qy[2][j][i],
                                            Q[1][l][k] - 0.5f*Qy[1][j][i],
                                            Q[3][l][k] - 0.5f*Qy[3][j][i]);
            const float4 Q_lr = make_float4(Q[0][l][k] + 0.5f*Qy[0][j][i],
                                            Q[2][l][k] + 0.5f*Qy[2][j][i],
                                            Q[1][l][k] + 0.5f*Qy[1][j][i],
                                            Q[3][l][k] + 0.5f*Qy[3][j][i]);
                                     
            //Evolve half a timestep (predictor step)
            const float4 Q_r_bar = Q_rl + dt_/(2.0f*dy_) * (F_func(Q_rl, gamma_) - F_func(Q_rr, gamma_));
            const float4 Q_l_bar = Q_lr + dt_/(2.0f*dy_) * (F_func(Q_ll, gamma_) - F_func(Q_lr, gamma_));
            
            // Compute flux based on prediction
            const float4 flux = make_float4(0.01, 0.01, 0.01, 0.01);//CentralUpwindFlux(Q_l_bar, Q_r_bar, gamma_);
            
            //Write to shared memory
            //Note that we here swap hu and hv back to the original
            G[0][j][i] = flux.x;
            G[1][j][i] = flux.z;
            G[2][j][i] = flux.y;
            G[3][j][i] = flux.w;
        }
    }
}




/**
  * This unsplit kernel computes the 2D numerical scheme with a TVD RK2 time integration scheme
  */
extern "C" {
__global__ void KP07DimsplitKernel(
        int nx_, int ny_,
        float dx_, float dy_, float dt_,
        float gamma_,
        
        float theta_,
        
        int step_,
        
        //Input h^n
        float* rho0_ptr_, int rho0_pitch_,
        float* rho_u0_ptr_, int rho_u0_pitch_,
        float* rho_v0_ptr_, int rho_v0_pitch_,
        float* E0_ptr_, int E0_pitch_,
        
        //Output h^{n+1}
        float* rho1_ptr_, int rho1_pitch_,
        float* rho_u1_ptr_, int rho_u1_pitch_,
        float* rho_v1_ptr_, int rho_v1_pitch_,
        float* E1_ptr_, int E1_pitch_) {
        
    const unsigned int w = BLOCK_WIDTH;
    const unsigned int h = BLOCK_HEIGHT;
    const unsigned int gc = 2;
        
    //Shared memory variables
    __shared__ float  Q[4][h+4][w+4];
    __shared__ float Qx[4][h+2][w+2];
    __shared__ float  F[4][h+1][w+1];
    
    
    
    //Read into shared memory
    readBlock<w, h, gc>(  rho0_ptr_,   rho0_pitch_, Q[0], nx_+2, ny_+2);
    readBlock<w, h, gc>(rho_u0_ptr_, rho_u0_pitch_, Q[1], nx_+2, ny_+2);
    readBlock<w, h, gc>(rho_v0_ptr_, rho_v0_pitch_, Q[2], nx_+2, ny_+2);
    readBlock<w, h, gc>(    E0_ptr_,     E0_pitch_, Q[3], nx_+2, ny_+2);
    __syncthreads();
    
    
    //Fix boundary conditions
    noFlowBoundary<w, h, gc,  1,  1>(Q[0], nx_, ny_);
    noFlowBoundary<w, h, gc, -1,  1>(Q[1], nx_, ny_);
    noFlowBoundary<w, h, gc,  1, -1>(Q[2], nx_, ny_);
    noFlowBoundary<w, h, gc,  1,  1>(Q[3], nx_, ny_);
    __syncthreads();
    
    
    //Step 0 => evolve x first, then y
    if (step_ == 0) {
        //Compute fluxes along the x axis and evolve
        minmodSlopeX(Q, Qx, theta_);
        __syncthreads();
        computeFluxF(Q, Qx, F, gamma_, dx_, dt_);
        __syncthreads();
        evolveF2(Q, F, nx_, ny_, dx_, dt_);
        __syncthreads();
        
        //Set boundary conditions
        noFlowBoundary<w, h, gc,  1,  1>(Q[0], nx_, ny_);
        noFlowBoundary<w, h, gc, -1,  1>(Q[1], nx_, ny_);
        noFlowBoundary<w, h, gc,  1, -1>(Q[2], nx_, ny_);
        noFlowBoundary<w, h, gc,  1,  1>(Q[3], nx_, ny_);
        __syncthreads();
        
        //Compute fluxes along the y axis and evolve
        minmodSlopeY(Q, Qx, theta_);
        __syncthreads();
        computeFluxG(Q, Qx, F, gamma_, dy_, dt_);
        __syncthreads();
        evolveG2(Q, F, nx_, ny_, dy_, dt_);
        __syncthreads();
    }
    //Step 1 => evolve y first, then x
    else {
        //Compute fluxes along the y axis and evolve
        minmodSlopeY(Q, Qx, theta_);
        __syncthreads();
        computeFluxG(Q, Qx, F, gamma_, dy_, dt_);
        __syncthreads();
        evolveG2(Q, F, nx_, ny_, dy_, dt_);
        __syncthreads();
        
        //Set boundary conditions
        noFlowBoundary<w, h, gc,  1,  1>(Q[0], nx_, ny_);
        noFlowBoundary<w, h, gc, -1,  1>(Q[1], nx_, ny_);
        noFlowBoundary<w, h, gc,  1, -1>(Q[2], nx_, ny_);
        noFlowBoundary<w, h, gc,  1,  1>(Q[3], nx_, ny_);
        __syncthreads();
        
        //Compute fluxes along the x axis and evolve
        minmodSlopeX(Q, Qx, theta_);
        __syncthreads();
        computeFluxF(Q, Qx, F, gamma_, dx_, dt_);
        __syncthreads();
        evolveF2(Q, F, nx_, ny_, dx_, dt_);
        __syncthreads();
    }
    
    
    // Write to main memory for all internal cells
    writeBlock<w, h, gc>(  rho1_ptr_,   rho1_pitch_, Q[0], nx_, ny_);
    writeBlock<w, h, gc>(rho_u1_ptr_, rho_u1_pitch_, Q[1], nx_, ny_);
    writeBlock<w, h, gc>(rho_v1_ptr_, rho_v1_pitch_, Q[2], nx_, ny_);
    writeBlock<w, h, gc>(    E1_ptr_,     E1_pitch_, Q[3], nx_, ny_);
}

} // extern "C"