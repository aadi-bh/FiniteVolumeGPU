#include "hip/hip_runtime.h"
 /*
This kernel implements the Central Upwind flux function to
solve the Euler equations 

Copyright (C) 2018  SINTEF Digital

This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/



#include "common.h"
#include "EulerCommon.h"
#include "limiters.h"


__device__
void computeFluxF(float Q[4][BLOCK_HEIGHT+4][BLOCK_WIDTH+4],
                  float Qx[4][BLOCK_HEIGHT+4][BLOCK_WIDTH+4],
                  float F[4][BLOCK_HEIGHT+4][BLOCK_WIDTH+4],
                  const float gamma_, const float dx_, const float dt_) {
    for (int j=threadIdx.y; j<BLOCK_HEIGHT+4; j+=BLOCK_HEIGHT) {
        for (int i=threadIdx.x+1; i<BLOCK_WIDTH+2; i+=BLOCK_WIDTH) {
            // Reconstruct point values of Q at the left and right hand side 
            // of the cell for both the left (i) and right (i+1) cell 
            const float4 Q_rl = make_float4(Q[0][j][i+1] - 0.5f*Qx[0][j][i+1],
                                            Q[1][j][i+1] - 0.5f*Qx[1][j][i+1],
                                            Q[2][j][i+1] - 0.5f*Qx[2][j][i+1],
                                            Q[3][j][i+1] - 0.5f*Qx[3][j][i+1]);
            const float4 Q_rr = make_float4(Q[0][j][i+1] + 0.5f*Qx[0][j][i+1],
                                            Q[1][j][i+1] + 0.5f*Qx[1][j][i+1],
                                            Q[2][j][i+1] + 0.5f*Qx[2][j][i+1],
                                            Q[3][j][i+1] + 0.5f*Qx[3][j][i+1]);

            const float4 Q_ll = make_float4(Q[0][j][i] - 0.5f*Qx[0][j][i],
                                            Q[1][j][i] - 0.5f*Qx[1][j][i],
                                            Q[2][j][i] - 0.5f*Qx[2][j][i],
                                            Q[3][j][i] - 0.5f*Qx[3][j][i]);
            const float4 Q_lr = make_float4(Q[0][j][i] + 0.5f*Qx[0][j][i],
                                            Q[1][j][i] + 0.5f*Qx[1][j][i],
                                            Q[2][j][i] + 0.5f*Qx[2][j][i],
                                            Q[3][j][i] + 0.5f*Qx[3][j][i]);


            //Evolve half a timestep (predictor step)
            const float4 Q_r_bar = Q_rl + dt_/(2.0f*dx_) * (F_func(Q_rl, gamma_) - F_func(Q_rr, gamma_));
            const float4 Q_l_bar = Q_lr + dt_/(2.0f*dx_) * (F_func(Q_ll, gamma_) - F_func(Q_lr, gamma_));

            // Compute flux based on prediction
            const float4 flux = CentralUpwindFlux(Q_l_bar, Q_r_bar, gamma_);
            
            //Write to shared memory
            F[0][j][i] = flux.x;
            F[1][j][i] = flux.y;
            F[2][j][i] = flux.z;
            F[3][j][i] = flux.w;
        }
    }
}

__device__
void computeFluxG(float Q[4][BLOCK_HEIGHT+4][BLOCK_WIDTH+4],
                  float Qy[4][BLOCK_HEIGHT+4][BLOCK_WIDTH+4],
                  float G[4][BLOCK_HEIGHT+4][BLOCK_WIDTH+4],
                  const float gamma_, const float dy_, const float dt_) {
    for (int j=threadIdx.y+1; j<BLOCK_HEIGHT+2; j+=BLOCK_HEIGHT) {
        for (int i=threadIdx.x; i<BLOCK_WIDTH+4; i+=BLOCK_WIDTH) {
            // Reconstruct point values of Q at the left and right hand side 
            // of the cell for both the left (i) and right (i+1) cell 
            //NOte that hu and hv are swapped ("transposing" the domain)!
            const float4 Q_rl = make_float4(Q[0][j+1][i] - 0.5f*Qy[0][j+1][i],
                                            Q[2][j+1][i] - 0.5f*Qy[2][j+1][i],
                                            Q[1][j+1][i] - 0.5f*Qy[1][j+1][i],
                                            Q[3][j+1][i] - 0.5f*Qy[3][j+1][i]);
            const float4 Q_rr = make_float4(Q[0][j+1][i] + 0.5f*Qy[0][j+1][i],
                                            Q[2][j+1][i] + 0.5f*Qy[2][j+1][i],
                                            Q[1][j+1][i] + 0.5f*Qy[1][j+1][i],
                                            Q[3][j+1][i] + 0.5f*Qy[3][j+1][i]);

            const float4 Q_ll = make_float4(Q[0][j][i] - 0.5f*Qy[0][j][i],
                                            Q[2][j][i] - 0.5f*Qy[2][j][i],
                                            Q[1][j][i] - 0.5f*Qy[1][j][i],
                                            Q[3][j][i] - 0.5f*Qy[3][j][i]);
            const float4 Q_lr = make_float4(Q[0][j][i] + 0.5f*Qy[0][j][i],
                                            Q[2][j][i] + 0.5f*Qy[2][j][i],
                                            Q[1][j][i] + 0.5f*Qy[1][j][i],
                                            Q[3][j][i] + 0.5f*Qy[3][j][i]);

            //Evolve half a timestep (predictor step)
            const float4 Q_r_bar = Q_rl + dt_/(2.0f*dy_) * (F_func(Q_rl, gamma_) - F_func(Q_rr, gamma_));
            const float4 Q_l_bar = Q_lr + dt_/(2.0f*dy_) * (F_func(Q_ll, gamma_) - F_func(Q_lr, gamma_));
            
            // Compute flux based on prediction
            const float4 flux = CentralUpwindFlux(Q_l_bar, Q_r_bar, gamma_);
            
            //Write to shared memory
            //Note that we here swap hu and hv back to the original
            G[0][j][i] = flux.x;
            G[1][j][i] = flux.z;
            G[2][j][i] = flux.y;
            G[3][j][i] = flux.w;
        }
    }
}



/**
  * This unsplit kernel computes the 2D numerical scheme with a TVD RK2 time integration scheme
  */
extern "C" {
__global__ void KP07DimsplitKernel(
        int nx_, int ny_,
        float dx_, float dy_, float dt_,
        float gamma_,
        
        float theta_,
        
        int step_,
        
        //Input h^n
        float* rho0_ptr_, int rho0_pitch_,
        float* rho_u0_ptr_, int rho_u0_pitch_,
        float* rho_v0_ptr_, int rho_v0_pitch_,
        float* E0_ptr_, int E0_pitch_,
        
        //Output h^{n+1}
        float* rho1_ptr_, int rho1_pitch_,
        float* rho_u1_ptr_, int rho_u1_pitch_,
        float* rho_v1_ptr_, int rho_v1_pitch_,
        float* E1_ptr_, int E1_pitch_) {
        
    const unsigned int w = BLOCK_WIDTH;
    const unsigned int h = BLOCK_HEIGHT;
    const unsigned int gc = 2;
    const unsigned int vars = 4;
        
    //Shared memory variables
    __shared__ float  Q[4][h+4][w+4];
    __shared__ float Qx[4][h+4][w+4];
    __shared__ float  F[4][h+4][w+4];
    
    
    
    //Read into shared memory
    readBlock<w, h, gc>(  rho0_ptr_,   rho0_pitch_, Q[0], nx_, ny_);
    readBlock<w, h, gc>(rho_u0_ptr_, rho_u0_pitch_, Q[1], nx_, ny_);
    readBlock<w, h, gc>(rho_v0_ptr_, rho_v0_pitch_, Q[2], nx_, ny_);
    readBlock<w, h, gc>(    E0_ptr_,     E0_pitch_, Q[3], nx_, ny_);
    __syncthreads();
    
    //Fix boundary conditions
    noFlowBoundary<w, h, gc,  1,  1>(Q[0], nx_, ny_);
    noFlowBoundary<w, h, gc,  1,  1>(Q[1], nx_, ny_);
    noFlowBoundary<w, h, gc,  1, -1>(Q[2], nx_, ny_);
    noFlowBoundary<w, h, gc,  1,  1>(Q[3], nx_, ny_);
    __syncthreads();


    //Step 0 => evolve x first, then y
    if (step_ == 0) {
        //Compute fluxes along the x axis and evolve
        minmodSlopeX<w, h, gc, vars>(Q, Qx, theta_);
        __syncthreads();

        computeFluxF(Q, Qx, F, gamma_, dx_, dt_);
        __syncthreads();

        evolveF<w, h, gc, vars>(Q, F, dx_, dt_);
        __syncthreads();

        //Set boundary conditions
        noFlowBoundary<w, h, gc,  1,  1>(Q[0], nx_, ny_);
        noFlowBoundary<w, h, gc,  1,  1>(Q[1], nx_, ny_);
        noFlowBoundary<w, h, gc,  1, -1>(Q[2], nx_, ny_);
        noFlowBoundary<w, h, gc,  1,  1>(Q[3], nx_, ny_);
        __syncthreads();

        //Compute fluxes along the y axis and evolve
        minmodSlopeY<w, h, gc, vars>(Q, Qx, theta_);
        __syncthreads();

        computeFluxG(Q, Qx, F, gamma_, dy_, dt_);
        __syncthreads();

        evolveG<w, h, gc, vars>(Q, F, dy_, dt_);
        __syncthreads();    

    }
    //Step 1 => evolve y first, then x
    else {
        //Compute fluxes along the y axis and evolve
        minmodSlopeY<w, h, gc, vars>(Q, Qx, theta_);
        __syncthreads();
  
        computeFluxG(Q, Qx, F, gamma_, dy_, dt_);
        __syncthreads();
  
        evolveG<w, h, gc, vars>(Q, F, dy_, dt_);
        __syncthreads();
  
        //Set boundary conditions
        noFlowBoundary<w, h, gc,  1,  1>(Q[0], nx_, ny_);
        noFlowBoundary<w, h, gc,  1,  1>(Q[1], nx_, ny_);
        noFlowBoundary<w, h, gc,  1, -1>(Q[2], nx_, ny_);
        noFlowBoundary<w, h, gc,  1,  1>(Q[3], nx_, ny_);
        __syncthreads();
        
        //Compute fluxes along the x axis and evolve
        minmodSlopeX<w, h, gc, vars>(Q, Qx, theta_);
        __syncthreads();

        computeFluxF(Q, Qx, F, gamma_, dx_, dt_);
        __syncthreads();

        evolveF<w, h, gc, vars>(Q, F, dx_, dt_);
        __syncthreads();
        
        //This is the RK2-part
        const int tx = threadIdx.x + gc;
        const int ty = threadIdx.y + gc;
        const float q1 = Q[0][ty][tx];
        const float q2 = Q[1][ty][tx];
        const float q3 = Q[2][ty][tx];
        const float q4 = Q[3][ty][tx];
        __syncthreads();
        
        readBlock<w, h, gc>(  rho1_ptr_,   rho1_pitch_, Q[0], nx_, ny_);
        readBlock<w, h, gc>(rho_u1_ptr_, rho_u1_pitch_, Q[1], nx_, ny_);
        readBlock<w, h, gc>(rho_v1_ptr_, rho_v1_pitch_, Q[2], nx_, ny_);
        readBlock<w, h, gc>(    E1_ptr_,     E1_pitch_, Q[3], nx_, ny_);
        __syncthreads();
        
        Q[0][ty][tx] = 0.5f*( Q[0][ty][tx] + q1 );
        Q[1][ty][tx] = 0.5f*( Q[1][ty][tx] + q2 );
        Q[2][ty][tx] = 0.5f*( Q[2][ty][tx] + q3 );
        Q[3][ty][tx] = 0.5f*( Q[3][ty][tx] + q4 );
    }

    
    // Write to main memory for all internal cells
    writeBlock<w, h, gc>(  rho1_ptr_,   rho1_pitch_, Q[0], nx_, ny_);
    writeBlock<w, h, gc>(rho_u1_ptr_, rho_u1_pitch_, Q[1], nx_, ny_);
    writeBlock<w, h, gc>(rho_v1_ptr_, rho_v1_pitch_, Q[2], nx_, ny_);
    writeBlock<w, h, gc>(    E1_ptr_,     E1_pitch_, Q[3], nx_, ny_);
}

} // extern "C"