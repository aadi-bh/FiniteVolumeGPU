#include "hip/hip_runtime.h"
 /*
This kernel implements the Central Upwind flux function to
solve the Euler equations 

Copyright (C) 2018  SINTEF Digital

This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/



#include "common.h"
#include "EulerCommon.h"
#include "limiters.h"


__device__
void computeFluxF(float Q[4][BLOCK_HEIGHT+4][BLOCK_WIDTH+4],
                  float Qx[4][BLOCK_HEIGHT+4][BLOCK_WIDTH+4],
                  float F[4][BLOCK_HEIGHT+4][BLOCK_WIDTH+4],
                  const float gamma_, const float dx_, const float dt_) {
    for (int j=threadIdx.y; j<BLOCK_HEIGHT+4; j+=BLOCK_HEIGHT) {
        for (int i=threadIdx.x+1; i<BLOCK_WIDTH+2; i+=BLOCK_WIDTH) {
            // Reconstruct point values of Q at the left and right hand side 
            // of the cell for both the left (i) and right (i+1) cell 
            const float4 Q_rl = make_float4(Q[0][j][i+1] - 0.5f*Qx[0][j][i+1],
                                            Q[1][j][i+1] - 0.5f*Qx[1][j][i+1],
                                            Q[2][j][i+1] - 0.5f*Qx[2][j][i+1],
                                            Q[3][j][i+1] - 0.5f*Qx[3][j][i+1]);
            const float4 Q_rr = make_float4(Q[0][j][i+1] + 0.5f*Qx[0][j][i+1],
                                            Q[1][j][i+1] + 0.5f*Qx[1][j][i+1],
                                            Q[2][j][i+1] + 0.5f*Qx[2][j][i+1],
                                            Q[3][j][i+1] + 0.5f*Qx[3][j][i+1]);

            const float4 Q_ll = make_float4(Q[0][j][i] - 0.5f*Qx[0][j][i],
                                            Q[1][j][i] - 0.5f*Qx[1][j][i],
                                            Q[2][j][i] - 0.5f*Qx[2][j][i],
                                            Q[3][j][i] - 0.5f*Qx[3][j][i]);
            const float4 Q_lr = make_float4(Q[0][j][i] + 0.5f*Qx[0][j][i],
                                            Q[1][j][i] + 0.5f*Qx[1][j][i],
                                            Q[2][j][i] + 0.5f*Qx[2][j][i],
                                            Q[3][j][i] + 0.5f*Qx[3][j][i]);


            //Evolve half a timestep (predictor step)
            const float4 Q_r_bar = Q_rl + dt_/(2.0f*dx_) * (F_func(Q_rl, gamma_) - F_func(Q_rr, gamma_));
            const float4 Q_l_bar = Q_lr + dt_/(2.0f*dx_) * (F_func(Q_ll, gamma_) - F_func(Q_lr, gamma_));

            // Compute flux based on prediction
            //const float4 flux = CentralUpwindFlux(Q_l_bar, Q_r_bar, gamma_);
            const float4 flux = HLL_flux(Q_l_bar, Q_r_bar, gamma_);
            
            //Write to shared memory
            F[0][j][i] = flux.x;
            F[1][j][i] = flux.y;
            F[2][j][i] = flux.z;
            F[3][j][i] = flux.w;
        }
    }
}

__device__
void computeFluxG(float Q[4][BLOCK_HEIGHT+4][BLOCK_WIDTH+4],
                  float Qy[4][BLOCK_HEIGHT+4][BLOCK_WIDTH+4],
                  float G[4][BLOCK_HEIGHT+4][BLOCK_WIDTH+4],
                  const float gamma_, const float dy_, const float dt_) {
    for (int j=threadIdx.y+1; j<BLOCK_HEIGHT+2; j+=BLOCK_HEIGHT) {
        for (int i=threadIdx.x; i<BLOCK_WIDTH+4; i+=BLOCK_WIDTH) {
            // Reconstruct point values of Q at the left and right hand side 
            // of the cell for both the left (i) and right (i+1) cell 
            //NOte that hu and hv are swapped ("transposing" the domain)!
            const float4 Q_rl = make_float4(Q[0][j+1][i] - 0.5f*Qy[0][j+1][i],
                                            Q[2][j+1][i] - 0.5f*Qy[2][j+1][i],
                                            Q[1][j+1][i] - 0.5f*Qy[1][j+1][i],
                                            Q[3][j+1][i] - 0.5f*Qy[3][j+1][i]);
            const float4 Q_rr = make_float4(Q[0][j+1][i] + 0.5f*Qy[0][j+1][i],
                                            Q[2][j+1][i] + 0.5f*Qy[2][j+1][i],
                                            Q[1][j+1][i] + 0.5f*Qy[1][j+1][i],
                                            Q[3][j+1][i] + 0.5f*Qy[3][j+1][i]);

            const float4 Q_ll = make_float4(Q[0][j][i] - 0.5f*Qy[0][j][i],
                                            Q[2][j][i] - 0.5f*Qy[2][j][i],
                                            Q[1][j][i] - 0.5f*Qy[1][j][i],
                                            Q[3][j][i] - 0.5f*Qy[3][j][i]);
            const float4 Q_lr = make_float4(Q[0][j][i] + 0.5f*Qy[0][j][i],
                                            Q[2][j][i] + 0.5f*Qy[2][j][i],
                                            Q[1][j][i] + 0.5f*Qy[1][j][i],
                                            Q[3][j][i] + 0.5f*Qy[3][j][i]);

            //Evolve half a timestep (predictor step)
            const float4 Q_r_bar = Q_rl + dt_/(2.0f*dy_) * (F_func(Q_rl, gamma_) - F_func(Q_rr, gamma_));
            const float4 Q_l_bar = Q_lr + dt_/(2.0f*dy_) * (F_func(Q_ll, gamma_) - F_func(Q_lr, gamma_));
            
            // Compute flux based on prediction
            const float4 flux = CentralUpwindFlux(Q_l_bar, Q_r_bar, gamma_);
            //const float4 flux = HLL_flux(Q_l_bar, Q_r_bar, gamma_);
            
            //Write to shared memory
            //Note that we here swap hu and hv back to the original
            G[0][j][i] = flux.x;
            G[1][j][i] = flux.z;
            G[2][j][i] = flux.y;
            G[3][j][i] = flux.w;
        }
    }
}



/**
  * This unsplit kernel computes the 2D numerical scheme with a TVD RK2 time integration scheme
  */
extern "C" {
    
__global__ void KP07DimsplitKernel(
        int nx_, int ny_,
        float dx_, float dy_, float dt_,
        float g_,
        float gamma_,
        
        float theta_,
        
        int step_,
        int boundary_conditions_,
        
        //Input h^n
        float* rho0_ptr_, int rho0_pitch_,
        float* rho_u0_ptr_, int rho_u0_pitch_,
        float* rho_v0_ptr_, int rho_v0_pitch_,
        float* E0_ptr_, int E0_pitch_,
        
        //Output h^{n+1}
        float* rho1_ptr_, int rho1_pitch_,
        float* rho_u1_ptr_, int rho_u1_pitch_,
        float* rho_v1_ptr_, int rho_v1_pitch_,
        float* E1_ptr_, int E1_pitch_, 
        
        //Output CFL
        float* cfl_,

        //Subarea of internal domain to compute
        int x0=0, int y0=0,
        int x1=0, int y1=0) {

    if(x1 == 0)
        x1 = nx_;

    if(y1 == 0)
        y1 = ny_;
    
    const unsigned int w = BLOCK_WIDTH;
    const unsigned int h = BLOCK_HEIGHT;
    const unsigned int gc_x = 2;
    const unsigned int gc_y = 2;
    const unsigned int vars = 4;
    
    //Shared memory variables
    __shared__ float  Q[4][h+2*gc_y][w+2*gc_x];
    __shared__ float Qx[4][h+2*gc_y][w+2*gc_x];
    __shared__ float  F[4][h+2*gc_y][w+2*gc_x];
    
    //Read into shared memory
    readBlock<w, h, gc_x, gc_y,  1,  1>(  rho0_ptr_,   rho0_pitch_, Q[0], nx_, ny_, boundary_conditions_, x0, y0, x1, y1);
    readBlock<w, h, gc_x, gc_y, -1,  1>(rho_u0_ptr_, rho_u0_pitch_, Q[1], nx_, ny_, boundary_conditions_, x0, y0, x1, y1);
    readBlock<w, h, gc_x, gc_y,  1, -1>(rho_v0_ptr_, rho_v0_pitch_, Q[2], nx_, ny_, boundary_conditions_, x0, y0, x1, y1);
    readBlock<w, h, gc_x, gc_y,  1,  1>(    E0_ptr_,     E0_pitch_, Q[3], nx_, ny_, boundary_conditions_, x0, y0, x1, y1);

    //Step 0 => evolve x first, then y
    if (step_ == 0) {
        //Compute fluxes along the x axis and evolve
        minmodSlopeX<w, h, gc_x, gc_y, vars>(Q, Qx, theta_);
        __syncthreads();
        computeFluxF(Q, Qx, F, gamma_, dx_, dt_);
        __syncthreads();
        evolveF<w, h, gc_x, gc_y, vars>(Q, F, dx_, dt_);
        __syncthreads();

        //Compute fluxes along the y axis and evolve
        minmodSlopeY<w, h, gc_x, gc_y, vars>(Q, Qx, theta_);
        __syncthreads();
        computeFluxG(Q, Qx, F, gamma_, dy_, dt_);
        __syncthreads();
        evolveG<w, h, gc_x, gc_y, vars>(Q, F, dy_, dt_);
        __syncthreads();    
        
        //Gravity source term
        if (g_ > 0.0f) {
            const int i = threadIdx.x + gc_x;
            const int j = threadIdx.y + gc_y;
            const float rho_v = Q[2][j][i];
            Q[2][j][i] -= g_*Q[0][j][i]*dt_;
            Q[3][j][i] -= g_*rho_v*dt_;
            __syncthreads();
        }
    }
    //Step 1 => evolve y first, then x
    else {
        //Compute fluxes along the y axis and evolve
        minmodSlopeY<w, h, gc_x, gc_y, vars>(Q, Qx, theta_);
        __syncthreads();
        computeFluxG(Q, Qx, F, gamma_, dy_, dt_);
        __syncthreads();
        evolveG<w, h, gc_x, gc_y, vars>(Q, F, dy_, dt_);
        __syncthreads();
        
        //Compute fluxes along the x axis and evolve
        minmodSlopeX<w, h, gc_x, gc_y, vars>(Q, Qx, theta_);
        __syncthreads();
        computeFluxF(Q, Qx, F, gamma_, dx_, dt_);
        __syncthreads();
        evolveF<w, h, gc_x, gc_y, vars>(Q, F, dx_, dt_);
        __syncthreads();
        
        //Gravity source term
        if (g_ > 0.0f) {
            const int i = threadIdx.x + gc_x;
            const int j = threadIdx.y + gc_y;
            const float rho_v = Q[2][j][i];
            Q[2][j][i] -= g_*Q[0][j][i]*dt_;
            Q[3][j][i] -= g_*rho_v*dt_;
            __syncthreads();
        }
    }

    
    // Write to main memory for all internal cells
    writeBlock<w, h, gc_x, gc_y>(  rho1_ptr_,   rho1_pitch_, Q[0], nx_, ny_, 0, 1, x0, y0, x1, y1);
    writeBlock<w, h, gc_x, gc_y>(rho_u1_ptr_, rho_u1_pitch_, Q[1], nx_, ny_, 0, 1, x0, y0, x1, y1);
    writeBlock<w, h, gc_x, gc_y>(rho_v1_ptr_, rho_v1_pitch_, Q[2], nx_, ny_, 0, 1, x0, y0, x1, y1);
    writeBlock<w, h, gc_x, gc_y>(    E1_ptr_,     E1_pitch_, Q[3], nx_, ny_, 0, 1, x0, y0, x1, y1);
    
    //Compute the CFL for this block
    if (cfl_ != NULL) {
        writeCfl<w, h, gc_x, gc_y, vars>(Q, F[0], nx_, ny_, dx_, dy_, gamma_, cfl_);
    }
}


} // extern "C"