#include "hip/hip_runtime.h"
/*
This GPU kernel implements the HLL flux

Copyright (C) 2016  SINTEF ICT

This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/



#include "common.h"
#include "SWECommon.h"





/**
  * Computes the flux along the x axis for all faces
  */
__device__
void computeFluxF(float Q[3][BLOCK_HEIGHT+2][BLOCK_WIDTH+2],
                  float F[3][BLOCK_HEIGHT+2][BLOCK_WIDTH+2],
                  const float g_) {
    for (int j=threadIdx.y; j<BLOCK_HEIGHT+2; j+=BLOCK_HEIGHT) {
        for (int i=threadIdx.x; i<BLOCK_WIDTH+1; i+=BLOCK_WIDTH) {
            // Q at interface from the right and left
            const float3 Q_r = make_float3(Q[0][j][i+1],
                                           Q[1][j][i+1],
                                           Q[2][j][i+1]);
            const float3 Q_l = make_float3(Q[0][j][i],
                                           Q[1][j][i],
                                           Q[2][j][i]);
                       
            // Computed flux
            const float3 flux = HLL_flux(Q_l, Q_r, g_);
            F[0][j][i] = flux.x;
            F[1][j][i] = flux.y;
            F[2][j][i] = flux.z;
        }
    }
}





/**
  * Computes the flux along the y axis for all faces
  */
__device__
void computeFluxG(float Q[3][BLOCK_HEIGHT+2][BLOCK_WIDTH+2],
                  float G[3][BLOCK_HEIGHT+2][BLOCK_WIDTH+2],
                  const float g_) {
    //Compute fluxes along the y axis
    for (int j=threadIdx.y; j<BLOCK_HEIGHT+1; j+=BLOCK_HEIGHT) {
        for (int i=threadIdx.x; i<BLOCK_WIDTH+2; i+=BLOCK_WIDTH) {
            // Q at interface from the right and left
            // Note that we swap hu and hv
            const float3 Q_r = make_float3(Q[0][j+1][i],
                                           Q[2][j+1][i],
                                           Q[1][j+1][i]);
            const float3 Q_l = make_float3(Q[0][j][i],
                                           Q[2][j][i],
                                           Q[1][j][i]);
                                       
            // Computed flux
            //Note that we here swap hu and hv back to the original
            const float3 flux = HLL_flux(Q_l, Q_r, g_);
            G[0][j][i] = flux.x;
            G[1][j][i] = flux.z;
            G[2][j][i] = flux.y;
        }
    }
}












extern "C" {
    
__global__ void HLLKernel(
        int nx_, int ny_,
        float dx_, float dy_, float dt_,
        float g_,
        
        //Input h^n
        float* h0_ptr_, int h0_pitch_,
        float* hu0_ptr_, int hu0_pitch_,
        float* hv0_ptr_, int hv0_pitch_,
        
        //Output h^{n+1}
        float* h1_ptr_, int h1_pitch_,
        float* hu1_ptr_, int hu1_pitch_,
        float* hv1_ptr_, int hv1_pitch_) {
    
    const unsigned int w = BLOCK_WIDTH;
    const unsigned int h = BLOCK_HEIGHT;
    const unsigned int gc = 1;
    const unsigned int vars = 3;
    
    //Shared memory variables
    __shared__ float Q[3][h+2][w+2];
    __shared__ float F[3][h+2][w+2];
    
    //Read into shared memory
    readBlock<w, h, gc>( h0_ptr_,  h0_pitch_, Q[0], nx_+2, ny_+2);
    readBlock<w, h, gc>(hu0_ptr_, hu0_pitch_, Q[1], nx_+2, ny_+2);
    readBlock<w, h, gc>(hv0_ptr_, hv0_pitch_, Q[2], nx_+2, ny_+2);
    __syncthreads();

    //Set boundary conditions
    noFlowBoundary<w, h, gc,  1,  1>(Q[0], nx_, ny_);
    noFlowBoundary<w, h, gc, -1,  1>(Q[1], nx_, ny_);
    noFlowBoundary<w, h, gc,  1, -1>(Q[2], nx_, ny_);
    __syncthreads();
    
    //Compute F flux
    computeFluxF(Q, F, g_);
    __syncthreads();
    
    evolveF<w, h, gc, vars>(Q, F, dx_, dt_);
    __syncthreads();
    
    //Set boundary conditions
    noFlowBoundary<w, h, gc,  1,  1>(Q[0], nx_, ny_);
    noFlowBoundary<w, h, gc, -1,  1>(Q[1], nx_, ny_);
    noFlowBoundary<w, h, gc,  1, -1>(Q[2], nx_, ny_);
    __syncthreads();
    
    //Compute G flux
    computeFluxG(Q, F, g_);
    __syncthreads();
    
    evolveG<w, h, gc, vars>(Q, F, dy_, dt_);
    __syncthreads();
    
    // Write to main memory for all internal cells
    writeBlock<w, h, gc>( h1_ptr_,  h1_pitch_, Q[0], nx_, ny_);
    writeBlock<w, h, gc>(hu1_ptr_, hu1_pitch_, Q[1], nx_, ny_);
    writeBlock<w, h, gc>(hv1_ptr_, hv1_pitch_, Q[2], nx_, ny_);
}

} // extern "C"