#include "hip/hip_runtime.h"
/*
This OpenCL kernel implements the Kurganov-Petrova numerical scheme 
for the shallow water equations, described in 
A. Kurganov & Guergana Petrova
A Second-Order Well-Balanced Positivity Preserving Central-Upwind
Scheme for the Saint-Venant System Communications in Mathematical
Sciences, 5 (2007), 133-160. 

Copyright (C) 2016  SINTEF ICT

This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/



#include "common.cu"
#include "SWECommon.cu"
#include "limiters.cu"
#include "fluxes/CentralUpwind.cu"


__device__
void computeFluxF(float Q[3][BLOCK_HEIGHT+4][BLOCK_WIDTH+4],
                  float Qx[3][BLOCK_HEIGHT+2][BLOCK_WIDTH+2],
                  float F[3][BLOCK_HEIGHT+1][BLOCK_WIDTH+1],
                  const float g_, const float dx_, const float dt_) {
    //Index of thread within block
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    
    {
        int j=ty;
        const int l = j + 2; //Skip ghost cells
        for (int i=tx; i<BLOCK_WIDTH+1; i+=BLOCK_WIDTH) {
            const int k = i + 1;
            // Reconstruct point values of Q at the left and right hand side 
            // of the cell for both the left (i) and right (i+1) cell 
            const float3 Q_rl = make_float3(Q[0][l][k+1] - 0.5f*Qx[0][j][i+1],
                                            Q[1][l][k+1] - 0.5f*Qx[1][j][i+1],
                                            Q[2][l][k+1] - 0.5f*Qx[2][j][i+1]);
            const float3 Q_rr = make_float3(Q[0][l][k+1] + 0.5f*Qx[0][j][i+1],
                                            Q[1][l][k+1] + 0.5f*Qx[1][j][i+1],
                                            Q[2][l][k+1] + 0.5f*Qx[2][j][i+1]);
                                         
            const float3 Q_ll = make_float3(Q[0][l][k] - 0.5f*Qx[0][j][i],
                                            Q[1][l][k] - 0.5f*Qx[1][j][i],
                                            Q[2][l][k] - 0.5f*Qx[2][j][i]);
            const float3 Q_lr = make_float3(Q[0][l][k] + 0.5f*Qx[0][j][i],
                                            Q[1][l][k] + 0.5f*Qx[1][j][i],
                                            Q[2][l][k] + 0.5f*Qx[2][j][i]);
                                    
            //Evolve half a timestep (predictor step)
            const float3 Q_r_bar = Q_rl + dt_/(2.0f*dx_) * (F_func(Q_rl, g_) - F_func(Q_rr, g_));
            const float3 Q_l_bar = Q_lr + dt_/(2.0f*dx_) * (F_func(Q_ll, g_) - F_func(Q_lr, g_));

            // Compute flux based on prediction
            const float3 flux = CentralUpwindFlux(Q_l_bar, Q_r_bar, g_);
            
            //Write to shared memory
            F[0][j][i] = flux.x;
            F[1][j][i] = flux.y;
            F[2][j][i] = flux.z;
        }
    }    
}

__device__
void computeFluxG(float Q[3][BLOCK_HEIGHT+4][BLOCK_WIDTH+4],
                  float Qy[3][BLOCK_HEIGHT+2][BLOCK_WIDTH+2],
                  float G[3][BLOCK_HEIGHT+1][BLOCK_WIDTH+1],
                  const float g_, const float dy_, const float dt_) {
    //Index of thread within block
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    
    for (int j=ty; j<BLOCK_HEIGHT+1; j+=BLOCK_HEIGHT) {
        const int l = j + 1;
        {
            int i=tx;
            const int k = i + 2; //Skip ghost cells
            // Reconstruct point values of Q at the left and right hand side 
            // of the cell for both the left (i) and right (i+1) cell 
            //NOte that hu and hv are swapped ("transposing" the domain)!
            const float3 Q_rl = make_float3(Q[0][l+1][k] - 0.5f*Qy[0][j+1][i],
                                            Q[2][l+1][k] - 0.5f*Qy[2][j+1][i],
                                            Q[1][l+1][k] - 0.5f*Qy[1][j+1][i]);
            const float3 Q_rr = make_float3(Q[0][l+1][k] + 0.5f*Qy[0][j+1][i],
                                            Q[2][l+1][k] + 0.5f*Qy[2][j+1][i],
                                            Q[1][l+1][k] + 0.5f*Qy[1][j+1][i]);
                                        
            const float3 Q_ll = make_float3(Q[0][l][k] - 0.5f*Qy[0][j][i],
                                            Q[2][l][k] - 0.5f*Qy[2][j][i],
                                            Q[1][l][k] - 0.5f*Qy[1][j][i]);
            const float3 Q_lr = make_float3(Q[0][l][k] + 0.5f*Qy[0][j][i],
                                            Q[2][l][k] + 0.5f*Qy[2][j][i],
                                            Q[1][l][k] + 0.5f*Qy[1][j][i]);
                                     
            //Evolve half a timestep (predictor step)
            const float3 Q_r_bar = Q_rl + dt_/(2.0f*dy_) * (F_func(Q_rl, g_) - F_func(Q_rr, g_));
            const float3 Q_l_bar = Q_lr + dt_/(2.0f*dy_) * (F_func(Q_ll, g_) - F_func(Q_lr, g_));
            
            // Compute flux based on prediction
            const float3 flux = CentralUpwindFlux(Q_l_bar, Q_r_bar, g_);
            
            //Write to shared memory
            //Note that we here swap hu and hv back to the original
            G[0][j][i] = flux.x;
            G[1][j][i] = flux.z;
            G[2][j][i] = flux.y;
        }
    }
}




/**
  * This unsplit kernel computes the 2D numerical scheme with a TVD RK2 time integration scheme
  */
extern "C" {
__global__ void KP07DimsplitKernel(
        int nx_, int ny_,
        float dx_, float dy_, float dt_,
        float g_,
        
        float theta_,
        
        int step_,
        
        //Input h^n
        float* h0_ptr_, int h0_pitch_,
        float* hu0_ptr_, int hu0_pitch_,
        float* hv0_ptr_, int hv0_pitch_,
        
        //Output h^{n+1}
        float* h1_ptr_, int h1_pitch_,
        float* hu1_ptr_, int hu1_pitch_,
        float* hv1_ptr_, int hv1_pitch_) {
        
        
    //Shared memory variables
    __shared__ float Q[3][BLOCK_HEIGHT+4][BLOCK_WIDTH+4];
    __shared__ float Qx[3][BLOCK_HEIGHT+2][BLOCK_WIDTH+2];
    __shared__ float F[3][BLOCK_HEIGHT+1][BLOCK_WIDTH+1];
    
    
    
    //Read into shared memory
    readBlock<BLOCK_WIDTH+4, BLOCK_HEIGHT+4>(h0_ptr_, h0_pitch_, Q[0], nx_+3, ny_+3);
    readBlock<BLOCK_WIDTH+4, BLOCK_HEIGHT+4>(hu0_ptr_, hu0_pitch_, Q[1], nx_+3, ny_+3);
    readBlock<BLOCK_WIDTH+4, BLOCK_HEIGHT+4>(hv0_ptr_, hv0_pitch_, Q[2], nx_+3, ny_+3);
    __syncthreads();
    
    
    //Fix boundary conditions
    noFlowBoundary2(Q, nx_, ny_);
    __syncthreads();
    
    
    
    //Step 0 => evolve x first, then y
    if (step_ == 0) {
        //Compute fluxes along the x axis and evolve
        minmodSlopeX(Q, Qx, theta_);
        __syncthreads();
        computeFluxF(Q, Qx, F, g_, dx_, dt_);
        __syncthreads();
        evolveF2(Q, F, nx_, ny_, dx_, dt_);
        __syncthreads();
        
        //Set boundary conditions
        noFlowBoundary2(Q, nx_, ny_);
        __syncthreads();
        
        //Compute fluxes along the y axis and evolve
        minmodSlopeY(Q, Qx, theta_);
        __syncthreads();
        computeFluxG(Q, Qx, F, g_, dy_, dt_);
        __syncthreads();
        evolveG2(Q, F, nx_, ny_, dy_, dt_);
        __syncthreads();
    }
    //Step 1 => evolve y first, then x
    else {
        //Compute fluxes along the y axis and evolve
        minmodSlopeY(Q, Qx, theta_);
        __syncthreads();
        computeFluxG(Q, Qx, F, g_, dy_, dt_);
        __syncthreads();
        evolveG2(Q, F, nx_, ny_, dy_, dt_);
        __syncthreads();
        
        //Set boundary conditions
        noFlowBoundary2(Q, nx_, ny_);
        __syncthreads();
        
        //Compute fluxes along the x axis and evolve
        minmodSlopeX(Q, Qx, theta_);
        __syncthreads();
        computeFluxF(Q, Qx, F, g_, dx_, dt_);
        __syncthreads();
        evolveF2(Q, F, nx_, ny_, dx_, dt_);
        __syncthreads();
    }
    
    
    // Write to main memory for all internal cells
    writeBlock2(h1_ptr_, h1_pitch_,
                hu1_ptr_, hu1_pitch_,
                hv1_ptr_, hv1_pitch_,
                Q, nx_, ny_);
}

} // extern "C"