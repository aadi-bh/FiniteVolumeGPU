#include "hip/hip_runtime.h"
/*
This file implements the Central upwind flux

Copyright (C) 2016, 2017, 2018 SINTEF ICT

This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/



/**
  * Central upwind flux function
  */
__device__ float3 CentralUpwindFlux(const float3 Qm, float3 Qp, const float g) {
    const float3 Fp = F_func(Qp, g);
    const float up = Qp.y / Qp.x;   // hu / h
    const float cp = sqrt(g*Qp.x); // sqrt(g*h)

    const float3 Fm = F_func(Qm, g);
    const float um = Qm.y / Qm.x;   // hu / h
    const float cm = sqrt(g*Qm.x); // sqrt(g*h)
    
    const float am = min(min(um-cm, up-cp), 0.0f); // largest negative wave speed
    const float ap = max(max(um+cm, up+cp), 0.0f); // largest positive wave speed
    
    return ((ap*Fm - am*Fp) + ap*am*(Qp-Qm))/(ap-am);
}

