#include "hip/hip_runtime.h"


/**
  * Lax-Friedrichs flux (Toro 2001, p 163)
  */
__device__ float3 LxF_1D_flux(const float3 Q_l, const float3 Q_r, const float g_, const float dx_, const float dt_) {
    const float3 F_l = F_func(Q_l, g_);
    const float3 F_r = F_func(Q_r, g_);
    
    return 0.5f*(F_l + F_r) + (dx_/(2.0f*dt_))*(Q_l - Q_r);
}



/**
  * Lax-Friedrichs extended to 2D
  */
__device__ float3 LxF_2D_flux(const float3 Q_l, const float3 Q_r, const float g_, const float dx_, const float dt_) {
    const float3 F_l = F_func(Q_l, g_);
    const float3 F_r = F_func(Q_r, g_);
    
    //Note numerical diffusion for 2D here (0.25)
    return 0.5f*(F_l + F_r) + (dx_/(4.0f*dt_))*(Q_l - Q_r);
}
