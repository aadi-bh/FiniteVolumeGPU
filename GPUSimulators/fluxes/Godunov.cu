#include "hip/hip_runtime.h"
/*
This file implements the Godunov flux

Copyright (C) 2016, 2017, 2018 SINTEF ICT

This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/




/**
  * Godunovs centered scheme (Toro 2001, p 165)
  */
__device__ float3 GodC_1D_flux(const float3 Q_l, const float3 Q_r, const float g_, const float dx_, const float dt_) {
    const float3 F_l = F_func(Q_l, g_);
    const float3 F_r = F_func(Q_r, g_);
    
    const float3 Q_godc = 0.5f*(Q_l + Q_r) + (dt_/dx_)*(F_l - F_r);
    
    return F_func(Q_godc, g_);
}
    


