#include "hip/hip_runtime.h"
/*
This OpenCL kernel implements the classical Lax-Friedrichs scheme
for the shallow water equations, with edge fluxes.

Copyright (C) 2016  SINTEF ICT

This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/


#include "Common.cu"
#include "SWECommon.cu"
#include "fluxes/LaxFriedrichs.cu"


/**
  * Computes the flux along the x axis for all faces
  */
template <int block_width, int block_height>
__device__ 
void computeFluxF(float Q[3][block_height+2][block_width+2],
                  float F[3][block_height][block_width+1],
                  const float g_, const float dx_, const float dt_) {
    //Index of thread within block
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    
    {
        const int j=ty;
        const int l = j + 1; //Skip ghost cells
        for (int i=tx; i<block_width+1; i+=block_width) {
            const int k = i;
            
            // Q at interface from the right and left
            const float3 Qp = make_float3(Q[0][l][k+1],
                                          Q[1][l][k+1],
                                          Q[2][l][k+1]);
            const float3 Qm = make_float3(Q[0][l][k],
                                          Q[1][l][k],
                                          Q[2][l][k]);
                                       
            // Computed flux
            const float3 flux = LxF_2D_flux(Qm, Qp, g_, dx_, dt_);
            F[0][j][i] = flux.x;
            F[1][j][i] = flux.y;
            F[2][j][i] = flux.z;
        }
    }
}


/**
  * Computes the flux along the y axis for all faces
  */ 
template <int block_width, int block_height>
__device__
void computeFluxG(float Q[3][block_height+2][block_width+2],
                  float G[3][block_height+1][block_width],
                  const float g_, const float dy_, const float dt_) {
    //Index of thread within block
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    
    for (int j=ty; j<block_height+1; j+=block_height) {
        const int l = j;
        {
            const int i=tx;
            const int k = i + 1; //Skip ghost cells
            
            // Q at interface from the right and left
            // Note that we swap hu and hv
            const float3 Qp = make_float3(Q[0][l+1][k],
                                          Q[2][l+1][k],
                                          Q[1][l+1][k]);
            const float3 Qm = make_float3(Q[0][l][k],
                                          Q[2][l][k],
                                          Q[1][l][k]);

            // Computed flux
            // Note that we swap back
            const float3 flux = LxF_2D_flux(Qm, Qp, g_, dy_, dt_);
            G[0][j][i] = flux.x;
            G[1][j][i] = flux.z;
            G[2][j][i] = flux.y;
        }
    }  
}



extern "C" {
__global__ 
void LxFKernel(
        int nx_, int ny_,
        float dx_, float dy_, float dt_,
        float g_,
        
        //Input h^n
        float* h0_ptr_, int h0_pitch_,
        float* hu0_ptr_, int hu0_pitch_,
        float* hv0_ptr_, int hv0_pitch_,
        
        //Output h^{n+1}
        float* h1_ptr_, int h1_pitch_,
        float* hu1_ptr_, int hu1_pitch_,
        float* hv1_ptr_, int hv1_pitch_) {
            
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    
    __shared__ float Q[3][BLOCK_HEIGHT+2][BLOCK_WIDTH+2];
    __shared__ float F[3][BLOCK_HEIGHT][BLOCK_WIDTH+1];
    __shared__ float G[3][BLOCK_HEIGHT+1][BLOCK_WIDTH];
    
    float* Q_ptr[3] = {h0_ptr_, hu0_ptr_, hv0_ptr_};
    int Q_pitch[3] = {h0_pitch_, hu0_pitch_, hv0_pitch_};
    
    readBlock<3, BLOCK_WIDTH+2, BLOCK_HEIGHT+2, BLOCK_WIDTH, BLOCK_HEIGHT>(Q_ptr, Q_pitch, Q, nx_+2, ny_+2);
    __syncthreads();
    
    //Set boundary conditions
    noFlowBoundary1(Q, nx_, ny_);
    __syncthreads();
    
    //Compute fluxes along the x and y axis
    computeFluxF<BLOCK_WIDTH, BLOCK_HEIGHT>(Q, F, g_, dx_, dt_);
    computeFluxG<BLOCK_WIDTH, BLOCK_HEIGHT>(Q, G, g_, dy_, dt_);
    __syncthreads();
    

    //Evolve for all cells
    const int i = tx + 1; //Skip local ghost cells, i.e., +1
    const int j = ty + 1;
    Q[0][j][i] += (F[0][ty][tx] - F[0][ty  ][tx+1]) * dt_ / dx_ 
                + (G[0][ty][tx] - G[0][ty+1][tx  ]) * dt_ / dy_;
    Q[1][j][i] += (F[1][ty][tx] - F[1][ty  ][tx+1]) * dt_ / dx_ 
                + (G[1][ty][tx] - G[1][ty+1][tx  ]) * dt_ / dy_;
    Q[2][j][i] += (F[2][ty][tx] - F[2][ty  ][tx+1]) * dt_ / dx_ 
                + (G[2][ty][tx] - G[2][ty+1][tx  ]) * dt_ / dy_;

    //Write to main memory
    writeBlock<BLOCK_WIDTH+2, BLOCK_HEIGHT+2, 1, 1>( h1_ptr_,  h1_pitch_, Q[0], nx_, ny_);
    writeBlock<BLOCK_WIDTH+2, BLOCK_HEIGHT+2, 1, 1>(hu1_ptr_, hu1_pitch_, Q[1], nx_, ny_);
    writeBlock<BLOCK_WIDTH+2, BLOCK_HEIGHT+2, 1, 1>(hv1_ptr_, hv1_pitch_, Q[2], nx_, ny_);
}

} // extern "C"

